#include "hip/hip_runtime.h"
#include "tracer.h"
#include <typeinfo>

vec3::vec3(){
	e[0] = 0;
	e[1] = 0;
	e[2] = 0;
}
vec3::vec3(float e0, float e1, float e2){
	e[0] = e0;
	e[1] = e1;
	e[2] = e2;
}
vec3::vec3(const vec3& v){
	e[0] = v.e[0];
	e[1] = v.e[1];
	e[2] = v.e[2];
}
float vec3::x() const{
	return e[0];
}
float vec3::y() const{
	return e[1];
}
float vec3::z() const{
	return e[2];
}
float vec3::r() const{
	return e[0];
}
float vec3::g() const{
	return e[1];
}
float vec3::b() const{
	return e[2];
}

vec3& vec3::operator=(const vec3& v){
	e[0] = v.e[0];
	e[1] = v.e[1];
	e[2] = v.e[2];
	return *this;
}
const vec3& vec3::operator+() const{
	return *this;
}
vec3 vec3::operator-() const{
	return vec3(-e[0], -e[1], -e[2]);
}
float vec3::operator[](int i) const{
	if(i < 3 && i > 0)
		return e[i];
	else return 0;
}
float& vec3::operator[](int i){
	return e[i];
}

vec3& vec3::operator+=(const vec3 &v2){
	e[0] += v2.e[0];
	e[1] += v2.e[1];
	e[2] += v2.e[2];
	return *this;
}
vec3& vec3::operator-=(const vec3 &v2){
	e[0] -= v2.e[0];
	e[1] -= v2.e[1];
	e[2] -= v2.e[2];
	return *this;
}
vec3& vec3::operator*=(const vec3 &v2){
	e[0] *= v2.e[0];
	e[1] *= v2.e[1];
	e[2] *= v2.e[2];
	return *this;}
vec3& vec3::operator/=(const vec3 &v2){
	e[0] /= v2.e[0];
	e[1] /= v2.e[1];
	e[2] /= v2.e[2];
	return *this;
}
vec3& vec3::operator*=(const float t){
	e[0] *= t;
	e[1] *= t;
	e[2] *= t;
	return *this;
}
vec3& vec3::operator/=(const float t){
	e[0] /= t;
	e[1] /= t;
	e[2] /= t;
	return *this;
}

float vec3::length() const{
	return sqrtf(e[0]*e[0] + e[1]*e[1] + e[2]*e[2]);
}
float vec3::squared_length() const{
	return e[0]*e[0] + e[1]*e[1] + e[2]*e[2];
}
void vec3::make_unit_vector(){
	float k = 1.0f / sqrt(e[0]*e[0] + e[1]*e[1] + e[2] * e[2]);
	e[0] *= k;
	e[0] *= k;
	e[0] *= k;
}

float vec3::dot(const vec3 &v2){
	return e[0]*v2.e[0] + e[1]*v2.e[1] + e[2]*v2.e[2];
}
vec3 vec3::cross(const vec3 &v2){
	return vec3(e[1]*v2.e[2] - e[2]*v2.e[1], (-(e[0]*v2.e[2] - e[2]*v2.e[0])), e[0]*v2.e[1] - e[1]*v2.e[0]);
}

istream& operator>>(istream &is, vec3 &t){
	is>>t.e[0]>>t.e[1]>>t.e[2];
	return is;
}
ostream& operator<<(ostream &os, vec3 &t){
	os<<t.e[0]<<' '<<t.e[1]<<' '<<t.e[2];
	return os;
}
vec3 operator+(const vec3 &v1, const vec3 &v2){
	return vec3(v1.e[0]+v2.e[0], v1.e[1]+v2.e[1], v1.e[2]+v2.e[2]);
}
vec3 operator-(const vec3 &v1, const vec3 &v2){
	return vec3(v1.e[0]-v2.e[0], v1.e[1]-v2.e[1], v1.e[2]-v2.e[2]);
}
vec3 operator*(const vec3 &v1, const vec3 &v2){
	return vec3(v1.e[0]*v2.e[0], v1.e[1]*v2.e[1], v1.e[2]*v2.e[2]);
}
vec3 operator/(const vec3 &v1, const vec3 &v2){
	return vec3(v1.e[0]/v2.e[0], v1.e[1]/v2.e[1], v1.e[2]/v2.e[2]);
}

vec3 operator*(const float t, const vec3 &v){
	return vec3(v.e[0]*t, v.e[1]*t, v.e[2]*t);
}
vec3 operator*(const vec3 &v, const float t){
	return vec3(v.e[0]*t, v.e[1]*t, v.e[2]*t);
}
vec3 operator/(const vec3 v, float t){
	return vec3(v.e[0]/t, v.e[1]/t, v.e[2]/t);
}

float dot(const vec3 &v1, const vec3 &v2){
	return v1.e[0]*v2.e[0] + v1.e[1]*v2.e[1] + v1.e[2]*v2.e[2];
}
vec3 cross(const vec3 &v1, const vec3 &v2){
	return vec3(v1.y()*v2.z() - v1.z()*v2.y(), (v1.z()*v2.x() - v1.x()*v2.z()), v1.x()*v2.y() - v1.y()*v2.x());
}

vec3 unit_vector(vec3 v){
	return v/v.length();
}

void vec3::set(float e0, float e1, float e2){
	e[0] = e0;
	e[1] = e1;
	e[2] = e2;
}

ray::ray(){}
ray::ray(const vec3& a, const vec3& b){
	A = a;
	B = b;
}
vec3 ray::origin() const{
	return A;
}
vec3 ray::direction() const{
	return B;
}
vec3 ray::p(float t) const{
	return A + t*B;
}
ray& ray::operator=(const ray& r){
	A = r.A;
	B = r.B;
	return *this;
}

sphere::sphere(){
	center = vec3(0,0,0);
	radius = 0;
}
sphere::sphere(vec3 cen, float r, material* m){
	center = cen;
	radius = r;
	mat = m;
}
bool sphere::hit(const ray& r, const float& tmin, float& tmax, hit_record& rec) const{
	vec3 oc = r.origin() - center;
	float a = dot(r.direction(), r.direction());
	float b = dot(oc, r.direction());
	float c = dot(oc, oc) - radius*radius;
	float discriminant = b*b-a*c;
	// printf("%f\n", radius);
	if (discriminant > 0){
		rec.mat = mat;
		float temp = (-b - sqrtf(b*b-a*c))/a;
		if (temp < tmax && temp > tmin){
			rec.t = temp;
			rec.p = r.p(rec.t);
			rec.normal = (rec.p - center) / radius;
			return true;
		}
		temp = (-b + sqrtf(b*b-a*c))/a;
		if(temp < tmax && temp > tmin){
			rec.t = temp;
			rec.p = r.p(rec.t);
			rec.normal = (rec.p - center) / radius;
			return true;
		}
	}
	return false;
}

hitable_list::hitable_list(){
	list = 0;
	list_size = 0;
}
hitable_list::hitable_list(hitable **list, int n){
	this->list = list;
	list_size = n;
}

__global__ void listHits(int n, int cluster, bool* anyHits, const ray* r, hitable** list, hit_record* temp_rec, float* dist, float tmin, float tmax, bool* finished){
	int index = threadIdx.x + blockDim.x*blockIdx.x;
	int curIndex = index * cluster;
	while(curIndex < n){
		for(int i = 0; i < cluster && curIndex+i < n; i++){
			if(list[curIndex+i]->hit(*r, tmin, tmax, temp_rec[curIndex+i])){
				anyHits[curIndex+i] = true;
				dist[curIndex+i] = temp_rec[curIndex+i].t;
			}
			else{
				anyHits[curIndex+i] = false;
			}
		}
		curIndex += gridDim.x*blockDim.x;
	}
	__syncthreads();
	if(index == 0){
		float max = tmax;
		for(int i = 0; i < n; i++){
			if(anyHits[i]){
				if(dist[i] < max){
					max = dist[i];
					anyHits[0] = true;
					dist[0] = max;
					temp_rec[0] = temp_rec[i];
				}
			}
		}
		*finished = true;
	}
}
// hitable** hitable_list::listPointer(){
// 	return d_list;
// }
__device__ bool hitable_list::hit(const ray& r, const float& tmin, float& tmax, hit_record& rec){//}, bool* d_hits, hit_record* d_recs, float* d_dmax) const{
	hit_record temp_rec;
	bool anyHits = false;
	// bool* finished = new bool;
	// *finished = false;
	float closest = tmax;
	// bool* d_hits = new bool[list_size];
	// hit_record* d_recs = new hit_record[list_size];
	// float* d_dmax = new float[list_size];
	// printf("%p, %d\n", this, this->list_size);
	// listHits<<<1, 256>>>(list_size, 1, d_hits, &r, list, d_recs, d_dmax, tmin, tmax, finished);
	// hipDeviceSynchronize();
	// while(!finished);
	// rec = d_recs[0];
	// anyHits = d_hits[0];

	for(int i = 0; i < list_size; i++){
		// printf("%d %d\n", i, list_size);
		// printf("%d %d\n", list_size, i);
		if(list[i]->hit(r, tmin, closest, temp_rec)){
			// printf("%f, %f, %f\n", r.direction().x(), r.direction().y(), r.direction().z());
			anyHits = true;
			closest = temp_rec.t;
			rec = temp_rec;
			// if(temp_rec.mat)
			// break;
		}
	}
	// delete[] d_hits;
	// delete[] d_recs;
	// delete[] d_dmax;
	return anyHits;
}

__device__ bool hitable_list::hit(const ray& r, const float& tmin, float& tmax, hit_record& rec, int index){
	if(index<list_size){
		hit_record temp_rec;
		// bool anyHits = false;
		// bool* finished = new bool;
		// *finished = false;
		float closest = tmax;
		if(list[index]->hit(r, tmin, closest, temp_rec)){
			closest = temp_rec.t;
			rec = temp_rec;
			return true;
		}
	}
	return false;
}

//apparently overrides of a parent's virtual functions don't work when the objects are instantiated on the host, instead must be instantiated through a backassward array of pointers and created entirely dynamically on the device
// void hitable_list::copyDevice(){
// 	hitable **h_list = new hitable*[list_size];
// 	// printf("%u\n", sizeof(hitable*));
// 	hipMalloc((void**)&d_list, sizeof(hitable*)*list_size);
// 	hipDeviceSynchronize();
// 	// printf("Done\n");
// 	for(int i = 0; i < list_size; i++){
// 		hitable* temp;
// 		// printf("%s %u\n", typeid(*list[i]).name(), sizeof(*list[i]));
// 		hipMalloc((void**)&temp, sizeof(*list[i]));
// 		hipDeviceSynchronize();
// 		hipMemcpy(temp, list[i], sizeof(*list[i]), hipMemcpyHostToDevice);
// 		hipDeviceSynchronize();
// 		h_list[i] = temp;
// 	}
// 	hipMemcpy(d_list, h_list, sizeof(hitable*)*list_size, hipMemcpyHostToDevice);
// 	hipDeviceSynchronize();
// 	// hitable_list* d_hlist;
// 	hipMalloc((void**)&d_world, sizeof(hitable_list));
// 	hipDeviceSynchronize();
// 	hitable **copy = list;
// 	list = d_list;
// 	// printf("%p %p %p\n", copy, list, h_list);
// 	// for(int i = 0; i < list_size; i++){
// 	// 	printf("copy: %p list: %p hlist: %p\n", copy[i], list[i], h_list[i]);
// 	// }
// 	hipMemcpy(d_world, this, sizeof(hitable_list), hipMemcpyHostToDevice);
// 	hipDeviceSynchronize();
// 	list = copy;
// }
__device__ vec3 random_in_unit_disk(hiprandState* state){
	// hiprandState state;
	// hiprand_init(1234, threadIdx.x+blockDim.x*blockIdx.x, 0, &state);
	vec3 p;
	do{
		p = 2.0f*vec3(hiprand_uniform(state), hiprand_uniform(state), 0) - vec3(1,1,0);
	}while(dot(p,p) >= 1.0f);
	return p;
}

vec3 random_in_unit_disk(mt19937 state){
	// hiprandState state;
	// hiprand_init(1234, threadIdx.x+blockDim.x*blockIdx.x, 0, &state);
	uniform_real_distribution<>dis(0,1);
	// printf("%f\n", state);
	vec3 p;
	do{
		p = 2.0f*vec3(dis(state), dis(state), 0) - vec3(1,1,0);
	}while(dot(p,p) >= 1.0f);
	// printf("exiting riud\n");
	return p;
}

camera::camera(){
	ulc = vec3(-2, 1, -1);
	horizontal = vec3(4, 0, 0);
	vertical = vec3(0,2,0);
	origin = vec3(0,0,0);
}

camera::camera(float vfov, float aspect){
	vfov *= CUDA_PI/180;
	float halfHeight = tanf(vfov/2);
	float halfWidth = aspect*halfHeight;
	ulc = vec3(-halfWidth, halfHeight, -1);
	horizontal = vec3(2*halfWidth, 0, 0);
	vertical = vec3(0,2*halfHeight,0);
	origin = vec3(0,0,0);
}
camera::camera(vec3 o, vec3 lookAt, vec3 vup, float vfov, float aspect){
	// vec3 u, v, w;
	lens_radius=0;
	vfov *= CUDA_PI/180;
	float halfHeight = tanf(vfov/2);
	float halfWidth = aspect*halfHeight;
	origin = o;
	w = unit_vector(o-lookAt);
	u = unit_vector(cross(vup, w));
	v = cross(w, u);
	ulc = vec3(-halfWidth, halfHeight, -1);
	ulc = origin - halfWidth*u + halfHeight*v - w;
	horizontal = 2*halfWidth*u;
	vertical = 2*halfHeight*v;
}
camera::camera(vec3 o, vec3 lookAt, vec3 vup, float vfov, float aspect, float aperture, float focus_dist){
	// vec3 u, v, w;
	lens_radius = aperture/2;
	vfov *= CUDA_PI/180;
	float halfHeight = tanf(vfov/2);
	float halfWidth = aspect*halfHeight;
	origin = o;
	w = unit_vector(o-lookAt);
	u = unit_vector(cross(vup, w));
	v = cross(w, u);
	// ulc = vec3(-halfWidth, halfHeight, -1);
	ulc = origin - halfWidth*focus_dist*u + halfHeight*focus_dist*v - focus_dist*w;
	horizontal = 2*halfWidth*u*focus_dist;
	vertical = 2*halfHeight*v*focus_dist;
}
__device__ void camera::get_ray(const float& s, const float& t, ray& r, hiprandState* state){
	vec3 rd;
	if(lens_radius > 0.001)
		rd = lens_radius * random_in_unit_disk(state);
	// printf("%f\n", v.y());
	vec3 offset = u*rd.x() + v*rd.y();
	r = ray(origin + offset, ulc+s*horizontal-t*vertical-origin-offset);
}

void camera::get_ray(const float& s, const float& t, ray& r, mt19937 state){
	vec3 rd;
	if(lens_radius > 0.001)
		rd = lens_radius * random_in_unit_disk(state);
	// printf("%f\n", v.y());
	vec3 offset = u*rd.x() + v*rd.y();
	r = ray(origin + offset, ulc+s*horizontal-t*vertical-origin-offset);
}

__host__ __device__ Face::Face(vec3 v1, vec3 v2, vec3 v3, vec3 t1, vec3 t2, vec3 t3, vec3 n1, vec3 n2, vec3 n3){
    verts[0] = v1;
    verts[1] = v2;
    verts[2] = v3;
    texts[0] = t1;
    texts[1] = t2;
    texts[2] = t3;
    normals[0] = n1;
    normals[1] = n2;
	normals[2] = n3;
	surfNorm = unit_vector(cross(verts[1]-verts[0], verts[2]-verts[1]));
	e[0] = verts[1] - verts[0];
    e[1] = verts[2] - verts[1];
    e[2] = verts[0] - verts[2];
	// vec3 avgNorms = unit_vector((n1 + n2 + n3)/3);
	// printf("verts: %f %f %f, %f %f %f, %f %f %f\n", verts[0].x(), verts[0].y(), verts[0].z(), verts[1].x(), verts[1].y(), verts[1].z(), verts[2].x(), verts[2].y(), verts[2].z());
	// if(avgNorms.x() != surfNorm.x() || avgNorms.y() != surfNorm.y() || avgNorms.z() != surfNorm.z())
	// printf("normals: %f %f %f vs %f %f %f\n", surfNorm.x(), surfNorm.y(), surfNorm.z(), avgNorms.x(), avgNorms.y(), avgNorms.z());
}

__host__ __device__ bool Face::hit(const ray& r, const float& t_min, float& t_max, hit_record& rec) const{//need to store non-ray derived values to reduce comp time
	float NdotDir = dot(surfNorm, r.direction());
	if(abs(NdotDir) < .001){
		// printf("parallel\n");
		return false;
	}
	float D = dot(surfNorm, verts[0]);
	
	float temp = -((dot(surfNorm, r.origin())-D)/NdotDir);
	// printf("%f\n", temp);
    vec3 p = (r.origin())+temp*(r.direction());
    // vec3 e[3];
    vec3 diff[3];
    
    diff[0] = p - verts[0];
    diff[1] = p - verts[1];
	diff[2] = p - verts[2];
	// e[0] = verts[1] - verts[0];
    // e[1] = verts[2] - verts[1];
    // e[2] = verts[0] - verts[2];
	// printf("%f %f %f\n", e[0].x(), e[1].x(), e[2].x());
	// printf("%f %f %f %f %f %f\n", r.A.x(), r.A.y(), r.A.z(), r.B.x(), r.B.y(), r.B.z());
	// if(p.length() < t_max)
		// printf("checking hit %f\n", temp);
	// else printf("t_max: %f\n", t_max);
	// printf("verts:\n%f %f %f\n%f %f %f\n%f %f %f\np:\n%f %f %f\n", verts[0].x(), verts[0].y(), verts[0].z(), verts[1].x(), verts[1].y(), verts[1].z(), verts[2].x(), verts[2].y(), verts[2].z(), p.x(), p.y(), p.z());

	for(int i = 0; i < 3; i++){
		// printf("%f\n",dot(surfNorm, cross(e[i], diff[i])));
        if(dot(surfNorm, cross(e[i], diff[i])) < 0){
			return false;
		}
	}
	// printf("ray is inside a triangle!\n");
	// printf("D: %f\nverts:\n%f %f %f\n%f %f %f\n%f %f %f\np: %f %f %f\nr.origin: %f %f %f\nr.direction: %f %f %f\n", D, verts[0].x(), verts[0].y(), verts[0].z(), verts[1].x(), verts[1].y(), verts[1].z(), verts[2].x(), verts[2].y(), verts[2].z(), p.x(), p.y(), p.z(), r.origin().x(), r.origin().y(), r.origin().z(), r.direction().x(), r.direction().y(), r.direction().z());

    if(temp < t_max && temp > t_min){
		t_max = temp;
		rec.mat = mat;
		rec.t = temp;
		rec.p = p;
		rec.normal = surfNorm;
		// printf("%f %f %f vs %f %f %f\n", surfNorm.x(), surfNorm.y(), surfNorm.z(), p.x(), p.y(), p.z());
		// printf("%f %f %f %f %f %f\n", r.A.x(), r.A.y(), r.A.z(), r.B.x(), r.B.y(), r.B.z());
		// printf("r.B: %f %f %f\n", r.B.x(), r.B.y(), r.B.z());
		// printf("p: %f %f %f\nv1: %f %f %f\nv2: %f %f %f\nv3: %f %f %f\n", p.x(), p.y(), p.z(), verts[0].x(), verts[0].y(), verts[0].z(), verts[1].x(), verts[1].y(), verts[1].z(), verts[2].x(), verts[2].y(), verts[2].z());
        return true;
    }
    return false;
}

lambertian::lambertian(const vec3& a){
	albedo = a;
	emitter = false;
}

__device__ bool lambertian::scatter(const ray& impacting, const hit_record& rec, vec3& att, ray& scattered, hiprandState* state) const{
	vec3 target = rec.p+rec.normal+random_in_unit_sphere(state);
	scattered = ray(rec.p, target-rec.p);
	att = albedo;
	return true;
}

metal::metal(const vec3& a, const float& f){
	emitter = false;
	albedo = a;
	if(f<1)
		fuzzy = f;
	else
		fuzzy = 1;
}

__device__ vec3 reflect(const vec3& v, const vec3& n){
	return v - 2*dot(v,n)*n;
}

__device__ bool metal::scatter(const ray& impacting, const hit_record& rec, vec3& att, ray& scattered, hiprandState* state) const{
	vec3 reflected = reflect(unit_vector(impacting.direction()), rec.normal);
	if(fuzzy >= 0.01)
		scattered = ray(rec.p, reflected + fuzzy*random_in_unit_sphere(state));
	else
		scattered = ray(rec.p, reflected);
	att = albedo;
	return (dot(scattered.direction(), rec.normal) > 0);
}

dielectric::dielectric(const float& i){
	ior = i;
	emitter = false;
}

__device__  bool dielectric::scatter(const ray& impacting, const hit_record& rec, vec3& att, ray& scattered, hiprandState* state) const{
	vec3 outward_normal;
	vec3 reflected = reflect(impacting.direction(), rec.normal);
	float ni_nt;
	att = vec3(1.0f, 1.0f, 1.0f);
	vec3 refracted;
	float reflect_prob;
	float cosine;
	float dotted = dot(impacting.direction(), rec.normal);
	if(dotted>0){//if normal and ray are facing same direction
		outward_normal = -rec.normal;
		ni_nt = ior;
		cosine = dotted/impacting.direction().length();
		cosine = sqrtf(1-ior*ior*(1-cosine*cosine));
	}
	else{
		outward_normal = rec.normal;
		ni_nt = 1.0f/ior;
		cosine = -dotted/impacting.direction().length();
	}
	if(refract(impacting.direction(), outward_normal, ni_nt, refracted)){
		reflect_prob = schlick(cosine, ior);
	}
	else{
		reflect_prob = 1;
	}
	if(hiprand_uniform(state) < reflect_prob){
		scattered = ray(rec.p, reflected);
	}
	else{
		scattered = ray(rec.p, refracted);
	}
	return true;
}

__device__ bool refract(const vec3& v, const vec3& n, float ni_nt, vec3& refracted){
	vec3 uv = unit_vector(v);
	float dt = dot(uv, n);
	float discriminant = 1.0f-ni_nt*ni_nt*(1.0f-dt*dt);
	if(discriminant > 0){
		refracted = ni_nt*(uv-n*dt) - n*sqrtf(discriminant);
		return true;
	}
	else return false;
}

__device__ float dielectric::schlick(const float& cosine, const float& indor) const{
	float r0 = (1-indor)/(1+indor);
	r0 = r0*r0;
	return r0 + (1-r0)*pow((1-cosine), 5);
}

__device__ vec3 random_in_unit_sphere(hiprandState* state){
	vec3 p;
	do {
		p = 2*vec3(hiprand_uniform(state),hiprand_uniform(state),hiprand_uniform(state)) - vec3(1,1,1);
	} while(p.squared_length() >= 1);
	return p;
}

// __device__ bool refract(const vec3&  v, const vec3& n, const float& ni_over_nt, vec3& refracted){
// 	vec3 uv = unit_vector(v);
// 	float dt = dot(uv, n);
// 	float discriminant = 1.0-ni_over_nt*ni_over_nt*(1-dt*dt);
// 	if(discriminant > 0){
// 		refracted = ni_over_nt*(uv - n*dt) - n*sqrtf(discriminant);
// 		return true;
// 	}
// 	else return false;
// }

// __device__ bool dielectric::scatter(const ray& r_in, const hit_record& rec, vec3& attenuation, ray& scattered, hiprandState* state) const{
// 	vec3 outward_normal;
// 	vec3 reflected = reflect(r_in.direction(), rec.normal);
// 	float ni_over_nt;
// 	attenuation = vec3(1.0f, 1.0f, 1.0f);
// 	vec3 refracted;
// 	if(dot(r_in.direction(), rec.normal)>0){
// 		outward_normal = -rec.normal;
// 		ni_over_nt = ref_idx;
// 	}
// 	else{
// 		outward_normal = rec.normal;
// 		ni_over_nt = 1.0f/ref_idx;
// 	}
// 	if(refract(r_in.direction(), outward_normal, ni_over_nt, refracted)){
// 		scattered = ray(rec.p, refracted);
// 	}
// 	else{
// 		scattered = ray(rec.p, reflected);
// 		return false;
// 	}
// 	return true;
// }

__device__ light::light(vec3 att){
	attenuation = att;
	emitter = true;
}

__device__ bool light::scatter(const ray& impacting, const hit_record& rec, vec3& att, ray& scattered, hiprandState* state) const{
	att = attenuation;
	scattered = impacting;
	printf("light!\n");
	return true;
}

__device__ hitable_list::hitable_list(OBJ **in, int n, int additional){

	list_size = 0;
	// printf("%d\n", n);
	for(int i = 0; i < n; i++){
		// printf("hl i: %d\n", i);
		list_size += in[i]->numFaces;
	}
	// printf("%d\n", list_size);
	list = new hitable*[list_size+additional];
	// int z = 0;
	// for(int i = 0; i < n; i++){
	// 	for(int j = 0; j < in[i]->numFaces; j++){
	// 		// printf("j: %d\n", j);
	// 		list[z] = new Face(in[i]->object[j], new light(vec3(4, 2, 2)));
	// 		// *list[z] = in[z];
	// 		z++;
	// 		if(z%10000 == 0)
	// 			printf("%d\n", z);
	// 	}
	// }
}

OBJ::OBJ(){
    points = 0;
    text = 0;
    normals = 0;
    numP = 0;
    numT = 0;
    numN = 0;
    numFaces = 0;
}

OBJ::OBJ(string fn){
    file = ifstream(fn);
    numP = 0;
    numT = 0;
    numN = 0;
    points = 0;
    text = 0;
    normals = 0;
	numFaces = 0;
	int i = 0;
    while(!file.eof() && !file.fail()){
        char line[1000];
        file.getline(line, 1000);
		parse(line);
		if(i%10000 == 0)
			printf("%d\n", i);
		i++;
	}
	file.close();
}

void OBJ::parse(char* line){
	// printf(line);
	// printf("\n");
    string buf = "";
    bool pp = false, tt = false, nn = false, newFace = false;
    float vec[3] = {0,0,0};
    int index = 0;
    int set[9];
    for(int i = 0; ; i++){
        if(line[i] == '#')
            break;
        if(line[i] == ' ' || line[i] == '\t' || line[i] == '\0'){
            if(!pp && !tt && !nn && !newFace && buf.compare("v") == 0){
                pp = true;
            }
            else if(!tt && !nn && !newFace && buf.compare("vt") == 0){
                tt = true;
            }
            else if(!nn && !newFace && buf.compare("vn") == 0){
                nn = true;
            }
            else if(!newFace && buf.compare("f") == 0){
                newFace = true;
            }
            else if((pp || tt || nn) && index < 3){
                vec[index] = stof(buf);
                index++;
            }
            else if(newFace && index < 3){
                int count = 0;
                string petiteBuf = "";
                for(int j = 0; j < buf.length()+1; j++){
                    if(buf[j] == '/' || buf[j] == '\0'){
						set[index*3 + count] = stoi(petiteBuf)-1;
						// printf("%d\n", set[index*3+count]);
						petiteBuf = "";
                        count++;
                    }
                    else{
                        petiteBuf += buf[j];
                    }
                }
                index++;
            }
            buf = "";
            if(line[i] == '\0')
                break;
            continue;
        }
        buf += line[i];
    }
    if(pp){
        append(points, numP, PBuf, vec3(vec[0], vec[1], vec[2]));
        // numP++;
    }
    else if(tt){
        append(text, numT, TBuf, vec3(vec[0], vec[1], 0.0f));
        // numT++;
    }
    else if(nn){
        append(normals, numN, NBuf, vec3(vec[0], vec[1], vec[2]));
        // numN++;
    }
    else if(newFace){
		// printf("%d: %f %f %f\n", set[0], points[set[0]].x(), points[set[0]].y(), points[set[0]].z());
        append(Face(points[set[0]], points[set[3]], points[set[6]], text[set[1]], text[set[4]], text[set[7]], normals[set[2]], normals[set[5]], normals[set[8]]));
		// exit(0);
	}
}

void OBJ::append(vec3*& list, int& size, int& bufSize, const vec3& item){
	if(size+1 > bufSize){
		vec3* temp = new vec3[bufSize+=1000];
		// printf("appending vectors\n");
		for(int i = 0; i < size; i++){
			temp[i] = list[i];
		}		
		if(size > 0)
			delete[] list;
		list = temp;
		// bufSize += 1000;
	}
	list[size] = item;
	size++;
}

void OBJ::append(const Face& item){
	if(numFaces + 1 > faceBuffer){
		Face* temp = new Face[faceBuffer+=1000];
		for(int i = 0; i < numFaces; i++){
			temp[i] = object[i];
		}
		// faceBuffer += 1000;
		if(numFaces > 0)
			delete[] object;
		object = temp;
	}
	object[numFaces] = item;
    numFaces++;
}

__host__ __device__ Face::Face(){
    verts[0] = vec3();
    verts[1] = vec3();
    verts[2] = vec3();
    texts[0] = vec3();
    texts[1] = vec3();
    texts[2] = vec3();
    normals[0] = vec3();
    normals[1] = vec3();
	normals[2] = vec3();
	e[0] = vec3();
	e[1] = vec3();
	e[2] = vec3();
}



__host__ __device__ Face& Face::operator=(const Face& in){
    verts[0] = in.verts[0];
    verts[1] = in.verts[1];
    verts[2] = in.verts[2];
    texts[0] = in.texts[0];
    texts[1] = in.texts[1];
    texts[2] = in.texts[2];
    normals[0] = in.normals[0];
    normals[1] = in.normals[1];
	normals[2] = in.normals[2];
	e[0] = verts[1] - verts[0];
    e[1] = verts[2] - verts[1];
    e[2] = verts[0] - verts[2];
	surfNorm = in.surfNorm;
	// surfNorm.make_unit_vector();
	// surfNorm = unit_vector(surfNorm);
	// vec3 temp = unit_vector(surfNorm);
	return *this;
}

OBJ* OBJ::copyToDevice(){
	// printf("entering ctd\n");
	gpuErrchk(hipDeviceSynchronize());
	// printf("synching\n");
	// printf("%d\n", sizeof(Face));
	// printf("%p\n", &numFaces);
	// printf("trying to malloc %d bytes\n", (sizeof(Face))*(numFaces));
	Face *d_faces, *oldFaces;
	gpuErrchk(hipMalloc((void**)&d_faces, sizeof(Face)*this->numFaces));
	// cout<<"mallocced faces\n";
    gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(d_faces, object, sizeof(Face)*this->numFaces, hipMemcpyHostToDevice));
	// cout<<"copied faces\n";
    oldFaces = object;
    object = d_faces;
    gpuErrchk(hipDeviceSynchronize());
    OBJ* d_obj;
    gpuErrchk(hipMalloc((void**)&d_obj, sizeof(OBJ)));
    gpuErrchk(hipMemcpy(d_obj, this, sizeof(OBJ), hipMemcpyHostToDevice));
    gpuErrchk(hipDeviceSynchronize());
    object = oldFaces;
    return d_obj;
}

__host__ __device__ bool OBJ::hit(const ray& r, const float& tmin, float& tmax, hit_record& rec) const{
    for(int i = 0; i < numFaces; i++){
        if(object[i].hit(r, tmin, tmax, rec))
            return true;
    }
    return false;
}

__host__ __device__ Face::Face(const Face& in){
    verts[0] = in.verts[0];
    verts[1] = in.verts[1];
    verts[2] = in.verts[2];
    texts[0] = in.texts[0];
    texts[1] = in.texts[1];
    texts[2] = in.texts[2];
    normals[0] = in.normals[0];
    normals[1] = in.normals[1];
	normals[2] = in.normals[2];
	e[0] = verts[1] - verts[0];
    e[1] = verts[2] - verts[1];
    e[2] = verts[0] - verts[2];
	// e[0] = in.e[0];
	// e[1] = in.e[1];
	// e[2] = in.e[2];
	surfNorm = in.surfNorm;
	mat = in.mat;
}

__host__ __device__ Face::Face(const Face& in, material* m){
	mat = m;
	verts[0] = in.verts[0];
    verts[1] = in.verts[1];
    verts[2] = in.verts[2];
    texts[0] = in.texts[0];
    texts[1] = in.texts[1];
    texts[2] = in.texts[2];
    normals[0] = in.normals[0];
    normals[1] = in.normals[1];
	normals[2] = in.normals[2];
	e[0] = verts[1] - verts[0];
    e[1] = verts[2] - verts[1];
    e[2] = verts[0] - verts[2];
	// printf("%f %f, %f %f, %f %f\n", verts[0].x(), in.verts[0].x(), verts[1].x(), in.verts[1].x(), verts[2].x(), in.verts[2].x());
	surfNorm = in.surfNorm;
}