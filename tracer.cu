#include "hip/hip_runtime.h"
#include "tracer.h"
#include <typeinfo>

vec3::vec3(){
	e[0] = 0;
	e[1] = 0;
	e[2] = 0;
}
vec3::vec3(float e0, float e1, float e2){
	e[0] = e0;
	e[1] = e1;
	e[2] = e2;
}
vec3::vec3(const vec3& v){
	e[0] = v.e[0];
	e[1] = v.e[1];
	e[2] = v.e[2];
}
float vec3::x() const{
	return e[0];
}
float vec3::y() const{
	return e[1];
}
float vec3::z() const{
	return e[2];
}
float vec3::r() const{
	return e[0];
}
float vec3::g() const{
	return e[1];
}
float vec3::b() const{
	return e[2];
}

vec3& vec3::operator=(const vec3& v){
	e[0] = v.e[0];
	e[1] = v.e[1];
	e[2] = v.e[2];
	return *this;
}
const vec3& vec3::operator+() const{
	return *this;
}
vec3 vec3::operator-() const{
	return vec3(-e[0], -e[1], -e[2]);
}
float vec3::operator[](int i) const{
	if(i < 3 && i > 0)
		return e[i];
	else return 0;
}
float& vec3::operator[](int i){
	return e[i];
}

vec3& vec3::operator+=(const vec3 &v2){
	e[0] += v2.e[0];
	e[1] += v2.e[1];
	e[2] += v2.e[2];
	return *this;
}
vec3& vec3::operator-=(const vec3 &v2){
	e[0] -= v2.e[0];
	e[1] -= v2.e[1];
	e[2] -= v2.e[2];
	return *this;
}
vec3& vec3::operator*=(const vec3 &v2){
	e[0] *= v2.e[0];
	e[1] *= v2.e[1];
	e[2] *= v2.e[2];
	return *this;}
vec3& vec3::operator/=(const vec3 &v2){
	e[0] /= v2.e[0];
	e[1] /= v2.e[1];
	e[2] /= v2.e[2];
	return *this;
}
vec3& vec3::operator*=(const float t){
	e[0] *= t;
	e[1] *= t;
	e[2] *= t;
	return *this;
}
vec3& vec3::operator/=(const float t){
	e[0] /= t;
	e[1] /= t;
	e[2] /= t;
	return *this;
}

float vec3::length() const{
	return sqrtf(e[0]*e[0] + e[1]*e[1] + e[2]*e[2]);
}
float vec3::squared_length() const{
	return e[0]*e[0] + e[1]*e[1] + e[2]*e[2];
}
void vec3::make_unit_vector(){
	float k = 1.0f / sqrt(e[0]*e[0] + e[1]*e[1] + e[2] * e[2]);
	e[0] *= k;
	e[0] *= k;
	e[0] *= k;
}

float vec3::dot(const vec3 &v2){
	return e[0]*v2.e[0] + e[1]*v2.e[1] + e[2]*v2.e[2];
}
vec3 vec3::cross(const vec3 &v2){
	return vec3(e[1]*v2.e[2] - e[2]*v2.e[1], (-(e[0]*v2.e[2] - e[2]*v2.e[0])), e[0]*v2.e[1] - e[1]*v2.e[0]);
}

istream& operator>>(istream &is, vec3 &t){
	is>>t.e[0]>>t.e[1]>>t.e[2];
	return is;
}
ostream& operator<<(ostream &os, vec3 &t){
	os<<t.e[0]<<' '<<t.e[1]<<' '<<t.e[2];
	return os;
}
vec3 operator+(const vec3 &v1, const vec3 &v2){
	return vec3(v1.e[0]+v2.e[0], v1.e[1]+v2.e[1], v1.e[2]+v2.e[2]);
}
vec3 operator-(const vec3 &v1, const vec3 &v2){
	return vec3(v1.e[0]-v2.e[0], v1.e[1]-v2.e[1], v1.e[2]-v2.e[2]);
}
vec3 operator*(const vec3 &v1, const vec3 &v2){
	return vec3(v1.e[0]*v2.e[0], v1.e[1]*v2.e[1], v1.e[2]*v2.e[2]);
}
vec3 operator/(const vec3 &v1, const vec3 &v2){
	return vec3(v1.e[0]/v2.e[0], v1.e[1]/v2.e[1], v1.e[2]/v2.e[2]);
}

vec3 operator*(const float t, const vec3 &v){
	return vec3(v.e[0]*t, v.e[1]*t, v.e[2]*t);
}
vec3 operator*(const vec3 &v, const float t){
	return vec3(v.e[0]*t, v.e[1]*t, v.e[2]*t);
}
vec3 operator/(const vec3 v, float t){
	return vec3(v.e[0]/t, v.e[1]/t, v.e[2]/t);
}

float dot(const vec3 &v1, const vec3 &v2){
	return v1.e[0]*v2.e[0] + v1.e[1]*v2.e[1] + v1.e[2]*v2.e[2];
}
vec3 cross(const vec3 &v1, const vec3 &v2){
	return vec3(v1.y()*v2.z() - v1.z()*v2.y(), (v1.z()*v2.x() - v1.x()*v2.z()), v1.x()*v2.y() - v1.y()*v2.x());
}

vec3 unit_vector(vec3 v){
	return v/v.length();
}

void vec3::set(float e0, float e1, float e2){
	e[0] = e0;
	e[1] = e1;
	e[2] = e2;
}

ray::ray(){}
ray::ray(const vec3& a, const vec3& b){
	A = a;
	B = b;
}
vec3 ray::origin() const{
	return A;
}
vec3 ray::direction() const{
	return B;
}
vec3 ray::p(float t) const{
	return A + t*B;
}
ray& ray::operator=(const ray& r){
	A = r.A;
	B = r.B;
	return *this;
}

sphere::sphere(){
	center = vec3(0,0,0);
	radius = 0;
}
sphere::sphere(vec3 cen, float r, material* m){
	center = cen;
	radius = r;
	mat = m;
}
__device__ bool sphere::hit(const ray& r, const float& tmin, float& tmax, hit_record& rec) const{
	vec3 oc = r.origin() - center;
	float a = dot(r.direction(), r.direction());
	float b = dot(oc, r.direction());
	float c = dot(oc, oc) - radius*radius;
	float discriminant = b*b-a*c;
	// printf("%f\n", radius);
	if (discriminant > 0){
		rec.mat = mat;
		float temp = (-b - sqrtf(b*b-a*c))/a;
		if (temp < tmax && temp > tmin){
			rec.t = temp;
			rec.p = r.p(rec.t);
			rec.normal = (rec.p - center) / radius;
			return true;
		}
		temp = (-b + sqrtf(b*b-a*c))/a;
		if(temp < tmax && temp > tmin){
			rec.t = temp;
			rec.p = r.p(rec.t);
			rec.normal = (rec.p - center) / radius;
			return true;
		}
	}
	return false;
}

hitable_list::hitable_list(){
	list = 0;
	list_size = 0;
}
hitable_list::hitable_list(hitable **list, int n){
	this->list = list;
	list_size = n;
}
hitable_list::hitable_list(int n){
	list_size = n;
	this->list = new hitable*[n];
}

__global__ void listHits(int n, int cluster, bool* anyHits, const ray* r, hitable** list, hit_record* temp_rec, float* dist, float tmin, float tmax, bool* finished){
	int index = threadIdx.x + blockDim.x*blockIdx.x;
	int curIndex = index * cluster;
	while(curIndex < n){
		for(int i = 0; i < cluster && curIndex+i < n; i++){
			if(list[curIndex+i]->hit(*r, tmin, tmax, temp_rec[curIndex+i])){
				anyHits[curIndex+i] = true;
				dist[curIndex+i] = temp_rec[curIndex+i].t;
			}
			else{
				anyHits[curIndex+i] = false;
			}
		}
		curIndex += gridDim.x*blockDim.x;
	}
	__syncthreads();
	if(index == 0){
		float max = tmax;
		for(int i = 0; i < n; i++){
			if(anyHits[i]){
				if(dist[i] < max){
					max = dist[i];
					anyHits[0] = true;
					dist[0] = max;
					temp_rec[0] = temp_rec[i];
				}
			}
		}
		*finished = true;
	}
}
// hitable** hitable_list::listPointer(){
// 	return d_list;
// }
__device__ bool hitable_list::hit(const ray& r, const float& tmin, float& tmax, hit_record& rec){//}, bool* d_hits, hit_record* d_recs, float* d_dmax) const{
	hit_record temp_rec;
	bool anyHits = false;
	// bool* finished = new bool;
	// *finished = false;
	float closest = tmax;
	// bool* d_hits = new bool[list_size];
	// hit_record* d_recs = new hit_record[list_size];
	// float* d_dmax = new float[list_size];
	// printf("%p, %d\n", this, this->list_size);
	// listHits<<<1, 256>>>(list_size, 1, d_hits, &r, list, d_recs, d_dmax, tmin, tmax, finished);
	// hipDeviceSynchronize();
	// while(!finished);
	// rec = d_recs[0];
	// anyHits = d_hits[0];

	for(int i = 0; i < list_size; i++){
		// printf("%d %d\n", i, list_size);
		// printf("%d %d\n", list_size, i);
		// printf("hh: %d %p %p %p %p\n", i, &r, &tmin, &closest, &temp_rec);
		if(list[i]->hit(r, tmin, closest, temp_rec)){
			// printf("%f, %f, %f\n", r.direction().x(), r.direction().y(), r.direction().z());
			anyHits = true;
			closest = temp_rec.t;
			rec = temp_rec;
			// if(temp_rec.mat)
			// break;
		}
	}
	// delete[] d_hits;
	// delete[] d_recs;
	// delete[] d_dmax;
	return anyHits;
}

__device__ bool hitable_list::hit(const ray& r, const float& tmin, float& tmax, hit_record& rec, int index){
	if(index<list_size){
		hit_record temp_rec;
		// bool anyHits = false;
		// bool* finished = new bool;
		// *finished = false;
		float closest = tmax;
		if(list[index]->hit(r, tmin, closest, temp_rec)){
			closest = temp_rec.t;
			rec = temp_rec;
			return true;
		}
	}
	return false;
}

//apparently overrides of a parent's virtual functions don't work when the objects are instantiated on the host, instead must be instantiated through a backassward array of pointers and created entirely dynamically on the device
// void hitable_list::copyDevice(){
// 	hitable **h_list = new hitable*[list_size];
// 	// printf("%u\n", sizeof(hitable*));
// 	hipMalloc((void**)&d_list, sizeof(hitable*)*list_size);
// 	hipDeviceSynchronize();
// 	// printf("Done\n");
// 	for(int i = 0; i < list_size; i++){
// 		hitable* temp;
// 		// printf("%s %u\n", typeid(*list[i]).name(), sizeof(*list[i]));
// 		hipMalloc((void**)&temp, sizeof(*list[i]));
// 		hipDeviceSynchronize();
// 		hipMemcpy(temp, list[i], sizeof(*list[i]), hipMemcpyHostToDevice);
// 		hipDeviceSynchronize();
// 		h_list[i] = temp;
// 	}
// 	hipMemcpy(d_list, h_list, sizeof(hitable*)*list_size, hipMemcpyHostToDevice);
// 	hipDeviceSynchronize();
// 	// hitable_list* d_hlist;
// 	hipMalloc((void**)&d_world, sizeof(hitable_list));
// 	hipDeviceSynchronize();
// 	hitable **copy = list;
// 	list = d_list;
// 	// printf("%p %p %p\n", copy, list, h_list);
// 	// for(int i = 0; i < list_size; i++){
// 	// 	printf("copy: %p list: %p hlist: %p\n", copy[i], list[i], h_list[i]);
// 	// }
// 	hipMemcpy(d_world, this, sizeof(hitable_list), hipMemcpyHostToDevice);
// 	hipDeviceSynchronize();
// 	list = copy;
// }
__device__ vec3 random_in_unit_disk(hiprandState* state){
	// hiprandState state;
	// hiprand_init(1234, threadIdx.x+blockDim.x*blockIdx.x, 0, &state);
	vec3 p;
	do{
		p = 2.0f*vec3(hiprand_uniform(state), hiprand_uniform(state), 0) - vec3(1,1,0);
	}while(dot(p,p) >= 1.0f);
	return p;
}

vec3 random_in_unit_disk(mt19937 state){
	// hiprandState state;
	// hiprand_init(1234, threadIdx.x+blockDim.x*blockIdx.x, 0, &state);
	uniform_real_distribution<>dis(0,1);
	// printf("%f\n", state);
	vec3 p;
	do{
		p = 2.0f*vec3(dis(state), dis(state), 0) - vec3(1,1,0);
	}while(dot(p,p) >= 1.0f);
	// printf("exiting riud\n");
	return p;
}

camera::camera(){
	ulc = vec3(-2, 1, -1);
	horizontal = vec3(4, 0, 0);
	vertical = vec3(0,2,0);
	origin = vec3(0,0,0);
}

camera::camera(float vfov, float aspect){
	vfov *= CUDA_PI/180;
	float halfHeight = tanf(vfov/2);
	float halfWidth = aspect*halfHeight;
	ulc = vec3(-halfWidth, halfHeight, -1);
	horizontal = vec3(2*halfWidth, 0, 0);
	vertical = vec3(0,2*halfHeight,0);
	origin = vec3(0,0,0);
}
camera::camera(vec3 o, vec3 lookAt, vec3 vup, float vfov, float aspect){
	// vec3 u, v, w;
	lens_radius=0;
	vfov *= CUDA_PI/180;
	float halfHeight = tanf(vfov/2);
	float halfWidth = aspect*halfHeight;
	origin = o;
	w = unit_vector(o-lookAt);
	u = unit_vector(cross(vup, w));
	v = cross(w, u);
	ulc = vec3(-halfWidth, halfHeight, -1);
	ulc = origin - halfWidth*u + halfHeight*v - w;
	horizontal = 2*halfWidth*u;
	vertical = 2*halfHeight*v;
}
camera::camera(vec3 o, vec3 lookAt, vec3 vup, float vfov, float aspect, float aperture, float focus_dist){
	// vec3 u, v, w;
	lens_radius = aperture/2;
	vfov *= CUDA_PI/180;
	float halfHeight = tanf(vfov/2);
	float halfWidth = aspect*halfHeight;
	origin = o;
	w = unit_vector(o-lookAt);
	u = unit_vector(cross(vup, w));
	v = cross(w, u);
	// ulc = vec3(-halfWidth, halfHeight, -1);
	ulc = origin - halfWidth*focus_dist*u + halfHeight*focus_dist*v - focus_dist*w;
	horizontal = 2*halfWidth*u*focus_dist;
	vertical = 2*halfHeight*v*focus_dist;
}
__device__ void camera::get_ray(const float& s, const float& t, ray& r, hiprandState* state){
	vec3 rd;
	if(lens_radius > 0.001)
		rd = lens_radius * random_in_unit_disk(state);
	// printf("%f\n", v.y());
	vec3 offset = u*rd.x() + v*rd.y();
	r = ray(origin + offset, ulc+s*horizontal-t*vertical-origin-offset);
}

void camera::get_ray(const float& s, const float& t, ray& r, mt19937 state){
	vec3 rd;
	if(lens_radius > 0.001)
		rd = lens_radius * random_in_unit_disk(state);
	// printf("%f\n", v.y());
	vec3 offset = u*rd.x() + v*rd.y();
	r = ray(origin + offset, ulc+s*horizontal-t*vertical-origin-offset);
}

__host__ __device__ Face::Face(vec3 v1, vec3 v2, vec3 v3, vec3 t1, vec3 t2, vec3 t3, vec3 n1, vec3 n2, vec3 n3){
    verts[0] = v1;
    verts[1] = v2;
    verts[2] = v3;
    texts[0] = t1;
    texts[1] = t2;
    texts[2] = t3;
    normals[0] = n1;
    normals[1] = n2;
	normals[2] = n3;
	surfNorm = unit_vector(cross(verts[1]-verts[0], verts[2]-verts[1]));
	e[0] = verts[1] - verts[0];
    e[1] = verts[2] - verts[1];
    e[2] = verts[0] - verts[2];
    // median.set((v1.x()+v2.x()+v3.x())/3, (v1.y()+v2.y()+v3.y())/3, (v1.z()+v2.z()+v3.z())/3);
    float x[3], y[3], z[3];
   
	x[0] = v1.x();
	y[0] = v1.y();
	z[0] = v1.z();
	x[1] = v2.x();
	y[1] = v2.y();
	z[1] = v2.z();
	x[2] = v3.x();
	y[2] = v3.y();
	z[2] = v3.z();

	for(int i = 0; i < 2; i++){
		for(int j = i; j < 3; j++){
			if(x[i] > x[j]){
				float temp = x[i];
				x[i] = x[j];
				x[j] = temp;
			}
			if(y[i] > y[j]){
				float temp = y[i];
				y[i] = y[j];
				y[j] = temp;
			}
			if(z[i] > z[j]){
				float temp = z[i];
				z[i] = z[j];
				z[j] = temp;
			}
		}
	}
	median.set(x[1], y[1], z[1]);

    for(int i =0; i < 3; i++){
		max[i] = FLT_MIN;
		min[i] = FLT_MAX;
		for(int j = 0; j < 3; j++){
			if(verts[j].e[i] > max[i])
				max[i] = verts[j].e[i];
			if(verts[j].e[i] < min[i])
				min[i] = verts[j].e[i];
		}
	}
    mat = nullptr;
	// vec3 avgNorms = unit_vector((n1 + n2 + n3)/3);
	// printf("verts: %f %f %f, %f %f %f, %f %f %f\n", verts[0].x(), verts[0].y(), verts[0].z(), verts[1].x(), verts[1].y(), verts[1].z(), verts[2].x(), verts[2].y(), verts[2].z());
	// if(avgNorms.x() != surfNorm.x() || avgNorms.y() != surfNorm.y() || avgNorms.z() != surfNorm.z())
	// printf("normals: %f %f %f vs %f %f %f\n", surfNorm.x(), surfNorm.y(), surfNorm.z(), avgNorms.x(), avgNorms.y(), avgNorms.z());
}



lambertian::lambertian(const vec3& a){
	albedo = a;
	emitter = false;
}

__device__ bool lambertian::scatter(const ray& impacting, const hit_record& rec, vec3& att, ray& scattered, hiprandState* state) const{
	vec3 target = rec.p+rec.normal+random_in_unit_sphere(state);
	scattered = ray(rec.p, target-rec.p);
	att = albedo;
	return true;
}

metal::metal(const vec3& a, const float& f){
	emitter = false;
	albedo = a;
	if(f<1)
		fuzzy = f;
	else
		fuzzy = 1;
}

__device__ vec3 reflect(const vec3& v, const vec3& n){
	return v - 2*dot(v,n)*n;
}

__device__ bool metal::scatter(const ray& impacting, const hit_record& rec, vec3& att, ray& scattered, hiprandState* state) const{
	vec3 reflected = reflect(unit_vector(impacting.direction()), rec.normal);
	if(fuzzy >= 0.01)
		scattered = ray(rec.p, reflected + fuzzy*random_in_unit_sphere(state));
	else
		scattered = ray(rec.p, reflected);
	att = albedo;
	return (dot(scattered.direction(), rec.normal) > 0);
}

dielectric::dielectric(const float& i){
	ior = i;
	emitter = false;
}

__device__  bool dielectric::scatter(const ray& impacting, const hit_record& rec, vec3& att, ray& scattered, hiprandState* state) const{
	vec3 outward_normal;
	vec3 reflected = reflect(impacting.direction(), rec.normal);
	float ni_nt;
	att = vec3(1.0f, 1.0f, 1.0f);
	vec3 refracted;
	float reflect_prob;
	float cosine;
	float dotted = dot(impacting.direction(), rec.normal);
	if(dotted>0){//if normal and ray are facing same direction
		outward_normal = -rec.normal;
		ni_nt = ior;
		cosine = dotted/impacting.direction().length();
		cosine = sqrtf(1-ior*ior*(1-cosine*cosine));
	}
	else{
		outward_normal = rec.normal;
		ni_nt = 1.0f/ior;
		cosine = -dotted/impacting.direction().length();
	}
	if(refract(impacting.direction(), outward_normal, ni_nt, refracted)){
		reflect_prob = schlick(cosine, ior);
	}
	else{
		reflect_prob = 1;
	}
	if(hiprand_uniform(state) < reflect_prob){
		scattered = ray(rec.p, reflected);
	}
	else{
		scattered = ray(rec.p, refracted);
	}
	return true;
}

__device__ bool refract(const vec3& v, const vec3& n, float ni_nt, vec3& refracted){
	vec3 uv = unit_vector(v);
	float dt = dot(uv, n);
	float discriminant = 1.0f-ni_nt*ni_nt*(1.0f-dt*dt);
	if(discriminant > 0){
		refracted = ni_nt*(uv-n*dt) - n*sqrtf(discriminant);
		return true;
	}
	else return false;
}

__device__ float dielectric::schlick(const float& cosine, const float& indor) const{
	float r0 = (1-indor)/(1+indor);
	r0 = r0*r0;
	return r0 + (1-r0)*pow((1-cosine), 5);
}

__device__ vec3 random_in_unit_sphere(hiprandState* state){
	vec3 p;
	// do {
		p = 2*vec3(hiprand_uniform(state),hiprand_uniform(state),hiprand_uniform(state)) - vec3(1,1,1);
	// } while(p.squared_length() >= 1);
	return unit_vector(p);
}

// __device__ bool refract(const vec3&  v, const vec3& n, const float& ni_over_nt, vec3& refracted){
// 	vec3 uv = unit_vector(v);
// 	float dt = dot(uv, n);
// 	float discriminant = 1.0-ni_over_nt*ni_over_nt*(1-dt*dt);
// 	if(discriminant > 0){
// 		refracted = ni_over_nt*(uv - n*dt) - n*sqrtf(discriminant);
// 		return true;
// 	}
// 	else return false;
// }

// __device__ bool dielectric::scatter(const ray& r_in, const hit_record& rec, vec3& attenuation, ray& scattered, hiprandState* state) const{
// 	vec3 outward_normal;
// 	vec3 reflected = reflect(r_in.direction(), rec.normal);
// 	float ni_over_nt;
// 	attenuation = vec3(1.0f, 1.0f, 1.0f);
// 	vec3 refracted;
// 	if(dot(r_in.direction(), rec.normal)>0){
// 		outward_normal = -rec.normal;
// 		ni_over_nt = ref_idx;
// 	}
// 	else{
// 		outward_normal = rec.normal;
// 		ni_over_nt = 1.0f/ref_idx;
// 	}
// 	if(refract(r_in.direction(), outward_normal, ni_over_nt, refracted)){
// 		scattered = ray(rec.p, refracted);
// 	}
// 	else{
// 		scattered = ray(rec.p, reflected);
// 		return false;
// 	}
// 	return true;
// }

__device__ light::light(vec3 att){
	attenuation = att;
	emitter = true;
}

__device__ bool light::scatter(const ray& impacting, const hit_record& rec, vec3& att, ray& scattered, hiprandState* state) const{
	att = attenuation;
	scattered = impacting;
	// printf("light!\n");
	return true;
}

__device__ hitable_list::hitable_list(OBJ **in, int n, int additional){

	list_size = 0;
	// printf("%d\n", n);
	for(int i = 0; i < n; i++){
		// printf("hl i: %d\n", i);
		list_size += in[i]->numFaces;
	}
	// printf("%d\n", list_size);
	list = new hitable*[list_size+additional];
	// int z = 0;
	// for(int i = 0; i < n; i++){
	// 	for(int j = 0; j < in[i]->numFaces; j++){
	// 		// printf("j: %d\n", j);
	// 		list[z] = new Face(in[i]->object[j], new light(vec3(4, 2, 2)));
	// 		// *list[z] = in[z];
	// 		z++;
	// 		if(z%10000 == 0)
	// 			printf("%d\n", z);
	// 	}
	// }
}

OBJ::OBJ(){
    points = 0;
    text = 0;
    normals = 0;
    numP = 0;
    numT = 0;
    numN = 0;
    numFaces = 0;
}

OBJ::OBJ(string fn){
    file = ifstream(fn);
    numP = 0;
    numT = 0;
    numN = 0;
    points = 0;
    text = 0;
    normals = 0;
	numFaces = 0;
	int i = 0;
    while(!file.eof() && !file.fail()){
        char line[1000];
        file.getline(line, 1000);
		parse(line);
		if(i%10000 == 0)
			printf("%d\n", i);
		i++;
	}
	file.close();
}

void OBJ::parse(char* line){
	// printf(line);
	// printf("\n");
    string buf = "";
    bool pp = false, tt = false, nn = false, newFace = false;
    float vec[3] = {0,0,0};
    int index = 0;
    int set[9];
    for(int i = 0; ; i++){
        if(line[i] == '#')
            break;
        if(line[i] == ' ' || line[i] == '\t' || line[i] == '\0'){
            if(!pp && !tt && !nn && !newFace && buf.compare("v") == 0){
                pp = true;
            }
            else if(!tt && !nn && !newFace && buf.compare("vt") == 0){
                tt = true;
            }
            else if(!nn && !newFace && buf.compare("vn") == 0){
                nn = true;
            }
            else if(!newFace && buf.compare("f") == 0){
                newFace = true;
            }
            else if((pp || tt || nn) && index < 3){
                vec[index] = stof(buf);
                index++;
            }
            else if(newFace && index < 3){
                int count = 0;
                string petiteBuf = "";
                for(int j = 0; j < buf.length()+1; j++){
                    if(buf[j] == '/' || buf[j] == '\0'){
						set[index*3 + count] = stoi(petiteBuf)-1;
						// printf("%d\n", set[index*3+count]);
						petiteBuf = "";
                        count++;
                    }
                    else{
                        petiteBuf += buf[j];
                    }
                }
                index++;
            }
            buf = "";
            if(line[i] == '\0')
                break;
            continue;
        }
        buf += line[i];
    }
    if(pp){
        append(points, numP, PBuf, vec3(vec[0], vec[1], vec[2]));
        // numP++;
    }
    else if(tt){
        append(text, numT, TBuf, vec3(vec[0], vec[1], 0.0f));
        // numT++;
    }
    else if(nn){
        append(normals, numN, NBuf, vec3(vec[0], vec[1], vec[2]));
        // numN++;
    }
    else if(newFace){
		// printf("%d: %f %f %f\n", set[0], points[set[0]].x(), points[set[0]].y(), points[set[0]].z());
        // append(Face(points[set[0]], points[set[3]], points[set[6]], text[set[1]], text[set[4]], text[set[7]], normals[set[2]], normals[set[5]], normals[set[8]]));
        append(Face(points[set[0]], points[set[3]], points[set[6]], vec3(), vec3(), vec3(), vec3(), vec3(), vec3()));
		// exit(0);
	}
}

void OBJ::append(vec3*& list, int& size, int& bufSize, const vec3& item){
	if(size+1 > bufSize){
		vec3* temp = new vec3[bufSize+=1000];
		// printf("appending vectors\n");
		for(int i = 0; i < size; i++){
			temp[i] = list[i];
		}		
		if(size > 0)
			delete[] list;
		list = temp;
		// bufSize += 1000;
	}
	list[size] = item;
	size++;
}

void OBJ::append(const Face& item){
	if(numFaces + 1 > faceBuffer){
		Face* temp = new Face[faceBuffer+=1000];
		for(int i = 0; i < numFaces; i++){
			temp[i] = object[i];
		}
		// faceBuffer += 1000;
		if(numFaces > 0)
			delete[] object;
		object = temp;
	}
	object[numFaces] = item;
    numFaces++;
}

__host__ __device__ Face::Face(){
    verts[0] = vec3();
    verts[1] = vec3();
    verts[2] = vec3();
    texts[0] = vec3();
    texts[1] = vec3();
    texts[2] = vec3();
    normals[0] = vec3();
    normals[1] = vec3();
	normals[2] = vec3();
	e[0] = vec3();
	e[1] = vec3();
	e[2] = vec3();
	median = vec3();
	mat = nullptr;
	min[0] = 0;
	min[1] = 0;
	min[2] = 0;
	max[0] = 0;
	max[1] = 0;
	max[2] = 0;
}



__host__ __device__ Face& Face::operator=(const Face& in){
    verts[0] = in.verts[0];
    verts[1] = in.verts[1];
    verts[2] = in.verts[2];
    texts[0] = in.texts[0];
    texts[1] = in.texts[1];
    texts[2] = in.texts[2];
    normals[0] = in.normals[0];
    normals[1] = in.normals[1];
	normals[2] = in.normals[2];
	e[0] = verts[1] - verts[0];
    e[1] = verts[2] - verts[1];
    e[2] = verts[0] - verts[2];
	surfNorm = in.surfNorm;
	mat = in.mat;
	median = in.median;
	min[0] = in.min[0];
	min[1] = in.min[1];
	min[2] = in.min[2];
	max[0] = in.max[0];
	max[1] = in.max[1];
	max[2] = in.max[2];
	// surfNorm.make_unit_vector();
	// surfNorm = unit_vector(surfNorm);
	// vec3 temp = unit_vector(surfNorm);
	return *this;
}

OBJ* OBJ::copyToDevice(){
	// printf("entering ctd\n");
	gpuErrchk(hipDeviceSynchronize());
	// printf("synching\n");
	// printf("%d\n", sizeof(Face));
	// printf("%p\n", &numFaces);
	// printf("trying to malloc %d bytes\n", (sizeof(Face))*(numFaces));
	Face *d_faces, *oldFaces;
	gpuErrchk(hipMalloc((void**)&d_faces, sizeof(Face)*this->numFaces));
	// cout<<"mallocced faces\n";
    gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(d_faces, object, sizeof(Face)*this->numFaces, hipMemcpyHostToDevice));
	// cout<<"copied faces\n";
    oldFaces = object;
    object = d_faces;
    gpuErrchk(hipDeviceSynchronize());
    OBJ* d_obj;
    gpuErrchk(hipMalloc((void**)&d_obj, sizeof(OBJ)));
    gpuErrchk(hipMemcpy(d_obj, this, sizeof(OBJ), hipMemcpyHostToDevice));
    gpuErrchk(hipDeviceSynchronize());
    object = oldFaces;
    return d_obj;
}

__device__ bool OBJ::hit(const ray& r, const float& tmin, float& tmax, hit_record& rec) const{
    for(int i = 0; i < numFaces; i++){
        if(object[i].hit(r, tmin, tmax, rec))
            return true;
    }
    return false;
}

__host__ __device__ Face::Face(const Face& in){
    verts[0] = in.verts[0];
    verts[1] = in.verts[1];
    verts[2] = in.verts[2];
    texts[0] = in.texts[0];
    texts[1] = in.texts[1];
    texts[2] = in.texts[2];
    normals[0] = in.normals[0];
    normals[1] = in.normals[1];
	normals[2] = in.normals[2];
	e[0] = verts[1] - verts[0];
    e[1] = verts[2] - verts[1];
    e[2] = verts[0] - verts[2];
	// e[0] = in.e[0];
	// e[1] = in.e[1];
	// e[2] = in.e[2];
	median = in.median;
	min[0] = in.min[0];
	min[1] = in.min[1];
	min[2] = in.min[2];
	max[0] = in.max[0];
	max[1] = in.max[1];
	max[2] = in.max[2];
	// surfNorm.make_uni
	surfNorm = in.surfNorm;
	mat = in.mat;
}

__host__ __device__ Face::Face(const Face& in, material* m){
	// printf("creating mat %p\n", m);
	mat = m;
	verts[0] = in.verts[0];
    verts[1] = in.verts[1];
    verts[2] = in.verts[2];
    texts[0] = in.texts[0];
    texts[1] = in.texts[1];
    texts[2] = in.texts[2];
    normals[0] = in.normals[0];
    normals[1] = in.normals[1];
	normals[2] = in.normals[2];
	e[0] = verts[1] - verts[0];
    e[1] = verts[2] - verts[1];
    e[2] = verts[0] - verts[2];
    min[0] = in.min[0];
	min[1] = in.min[1];
	min[2] = in.min[2];
	max[0] = in.max[0];
	max[1] = in.max[1];
	max[2] = in.max[2];
	// surfNorm.make_uni
    median = in.median;
	// printf("%f %f, %f %f, %f %f\n", verts[0].x(), in.verts[0].x(), verts[1].x(), in.verts[1].x(), verts[2].x(), in.verts[2].x());
	surfNorm = in.surfNorm;
}

__device__ sss::sss(material* surf, const float& d, const vec3& internal){
	attenuation = internal;
	depth = d;
	surface = surf;
}

__device__ bool sss::scatter(const ray& impacting, const hit_record& rec, vec3& att, ray& scattered, hiprandState* state) const{
	if(dot(unit_vector(impacting.direction()), unit_vector(rec.normal)) > 0){
		// printf("inside out %f %f %f, %f %f %f\n", rec.p.x(), rec.p.y(), rec.p.z(), impacting.direction().x(), impacting.direction().y(), impacting.direction().z());
		vec3 temp = impacting.direction();
		vec3 tAtt;
		ray tScattered;
		surface->scatter(impacting, rec, tAtt, tScattered, state);
		do{
			temp = random_in_unit_sphere(state);
		}	while(dot(temp, rec.normal) <= 0);
		scattered = ray(rec.p, impacting.direction() + temp);
		float l = depth/(rec.p - impacting.origin()).length();
		if(l > 1)
			l = 1;
		att = vec3(l, l, l);
		att = tAtt;
		att /= 2;
		return true;
		// return surface->scatter(impacting, rec, att, scattered, state);
	}
	else if(hiprand_uniform(state) > 0.5f){//determines if reflecting off surface
		return surface->scatter(impacting, rec, att, scattered, state);
	}
	else{//or going inside
		// vec3 internalTarget = (dot(rec.p, -rec.normal)/rec.normal.length()) * unit_vector(-rec.normal);
		vec3 temp = impacting.direction();
		do{
			temp = random_in_unit_sphere(state);
		}	while(dot(temp, rec.normal) >= 0);
		// printf("%f %f %f, %f %f %f\n", rec.normal.x(), rec.normal.y(), rec.normal.z(), temp.x(), temp.y(), temp.z());
		scattered = ray(rec.p, impacting.direction() + temp);
		att = attenuation;
		return true;
	}
}

__device__ TreeNode::TreeNode(){
	parent = nullptr;
	obj = nullptr;
	dim = 0;
	r = nullptr;
	l = nullptr;
	within = 0;
	contained = nullptr;
	min[0] = FLT_MAX;
	min[1] = FLT_MAX;
	min[2] = FLT_MAX;
	max[0] = FLT_MIN;
	max[1] = FLT_MIN;
	max[2] = FLT_MIN;
}
__device__ TreeNode::TreeNode(Face* in, TreeNode* par){
	// printf("inserted\n");
	// obj = nullptr;
	l = r = nullptr;
	parent = par;
	min[0] = in->min[0];
	min[1] = in->min[1];
	min[2] = in->min[2];
	max[0] = in->max[0];
	max[1] = in->max[1];
	max[2] = in->max[2];
	within = 0;
	contained = nullptr;
	// surfNorm.make_uni
	if(par != nullptr)
		dim = parent->dim<2 ? par->dim+1 : 0;
	else{
		dim = 0;
	}
	median = in->median;
	p = median[dim];	//note that this is mean value, may have to use median (probably won't matter)
	obj = in;
}
__device__ bool TreeNode::hit(const ray& r, const float& tmin, float& tmax, hit_record& rec) const{
	if(obj != nullptr){
		return obj->hit(r, tmin, tmax, rec);
	}
	return false;
}

__device__ bool TreeNode::withinBB(const vec3& p){
	for(int i = 0; i < 2; i++){
		if(p.e[i] > max[i] || p.e[i] < min[i])
			return false;
	}
	return true;
}

__device__ TriTree::TriTree(){
	numNodes = 0;
	head = nullptr;
}
__device__ void TriTree::insert(Face* in){ //TODO: Need to preprocess total triangles, create bounding box at each node level and split, then create leaf nodes. This takes care of order of insertion problem	
	TreeNode* cur = head, *prev = nullptr;
	numNodes++;
	while(cur != nullptr){
		cur->within = 0;
		// cur->medSum += in->median;
		// cur->median = cur->medSum/cur->within;
		for(int i = 0; i < 3; i++){
			if(in->max[i] > cur->max[i])
				cur->max[i] = in->max[i];
			if(in->min[i] < cur->min[i])
				cur->min[i] = in->min[i];
		}
		prev = cur;
		if(in->median.e[cur->dim] < cur->p){
			cur = cur->l;
			if(cur == nullptr){
				prev->l = new TreeNode(in, prev);
				break;
			}
		}
		else{
			cur = cur->r;
			if(cur == nullptr){
				prev->r = new TreeNode(in, prev);
				break;
			}
		}
	}
	if(head == nullptr){
		head = new TreeNode(in, nullptr);
	}
}



__device__ bool Face::hit(const ray& r, const float& t_min, float& t_max, hit_record& rec) const{//need to store non-ray derived values to reduce comp time
	vec3 one = vec3(1,1,1);
	for(int i = 0; i < 3; i++){
		if(dot(verts[i], r.direction()) - dot(one, t_max*r.direction()) > 0)
			break;
		if(i == 2)
			return false;
	}
	float NdotDir = dot(surfNorm, r.direction());
	if(abs(NdotDir) < .001){
		return false;
	}
	float D = dot(surfNorm, verts[0]);
	
	float temp = -((dot(surfNorm, r.origin())-D)/NdotDir);
    vec3 p = (r.origin())+temp*(r.direction());
    vec3 diff[3];
    
    diff[0] = p - verts[0];
    diff[1] = p - verts[1];
	diff[2] = p - verts[2];

	for(int i = 0; i < 3; i++){
        if(dot(surfNorm, cross(e[i], diff[i])) < 0){
			return false;
		}
	}

    if(temp < t_max && temp > t_min){
		t_max = temp;
		rec.mat = mat;
		rec.t = temp;
		rec.p = p;
		rec.normal = surfNorm;
        return true;
    }
    return false;
}

__device__ bool TriTree::positionOnPlane(const ray& r, TreeNode* n, vec3& poi) const {	//TODO: fix error where half the triangles don't render at all (has to do with left/right in plane for each tri)
	vec3 planeNormal;//, planePos;
	int dimension = n->dim;
	dimension = dimension == 0 ? 2 : dimension-1;
	planeNormal.e[dimension] = 1;
	float denom = dot(r.direction(), planeNormal);

	if(abs(denom) < 0.0001f)
		return false;

	vec3 vAdjusted = r.direction()*(-dot((n->median - r.origin()), planeNormal)/denom);
	poi = r.origin() - vAdjusted;
	return true;
}

__device__ bool TriTree::hit(const ray& r, const float& tmin, float& tmax, hit_record& rec) const{	//Need to fix math/logic error in positionOnPlane
	TreeNode* cur = head;//, *t = nullptr;
	// printf("numNodes: %d\n", numNodes);
	TreeNode** stack = new TreeNode*[numNodes];
	int stackSize = 0;
	bool anyhit = false;
	vec3 pos;
	hit_record temprec;
	float closest = tmax;
	bool *position = new bool[numNodes];
	do{
		while(cur != nullptr){
			stack[stackSize++] = cur;
			if(position[stackSize-1] = positionOnPlane(r, cur, pos)){
				if(cur->l == nullptr && cur->r == nullptr){
					// printf("within: %d\n", cur->within);
					for(int i = 0; i < cur->within; i++){
						if(cur->contained[i]->hit(r, tmin, closest, temprec)){
							if(closest < tmax){
								rec = temprec;
								tmax = closest;
								anyhit = true;
							}
						}
					}
				}
				if(pos.e[cur->dim] < cur->median[cur->dim]){
					cur = cur->l;
				}
				else if(pos.e[cur->dim] >= cur->median[cur->dim]){
					cur = cur->r;
				}
			}
			else{
				cur = cur->l;
			}
		}
		while(stackSize > 0 && (position[stackSize-1] || stack[stackSize-1]->r == nullptr)){
			stackSize--;
		}
		if(stackSize > 0){
			cur = stack[stackSize-1]->r;
			stackSize--;
		}
	}	while(stackSize > 0);
	delete[] stack;
	delete[] position;
	return anyhit;
}

__device__ void sortInsertion(int max, float* mx, float* my, float* mz, const vec3& med){
	const float *median = med.e;

	for(int i = 0; i <= max; i++){
		if(i == max){
			mx[i] = median[0];
		}
		else{
			if(median[0] < mx[i]){
				for(int r = max; r > i; r--){
					mx[r] = mx[r-1];
				}
				mx[i] = median[0];
				break;
			}
		}
	}
	for(int i = 0; i <= max; i++){
		if(i == max){
			my[i] = median[1];
		}
		else{
			if(median[1] < my[i]){
				for(int r = max; r > i; r--){
					mx[r] = my[r-1];
				}
				my[i] = median[1];
				break;
			}
		}
	}
	for(int i = 0; i <= max; i++){
		if(i == max){
			mz[i] = median[2];
		}
		else{
			if(median[2] < mz[i]){
				for(int r = max; r > i; r--){
					mz[r] = mz[r-1];
				}
				mz[i] = median[2];
				break;
			}
		}
	}
}

__device__ TriTree* OBJ::toTree(){
	int list_size = numFaces;
	TreeNode** stack = new TreeNode*[list_size];
	unsigned int stackSize = 0, maxSize = 0;
	stack[stackSize++] = new TreeNode;
	stack[stackSize-1]->contained = new Face*[list_size];
	float *mx = new float[list_size], *my = new float[list_size], *mz = new float[list_size];
	
	for(int i = 0; i < list_size; i++){
		// stack[stackSize-1]->contained[i] = new Face(object[i], new lambertian(vec3(0.0f, 0.2f, 0.0f)));
		stack[stackSize-1]->contained[i] = new Face(object[i], new sss( new lambertian(vec3(0.0f, 0.2f, 0.0f)), 0.05f, vec3(1.0f, 0.25f, 0.2f)));

		sortInsertion(i, mx, my, mz, stack[stackSize-1]->contained[i]->median);
	}

	if(list_size % 2 != 0){
		stack[stackSize-1]->median.set(mx[list_size/2], my[list_size/2], mz[list_size/2]);
	}
	else{
		stack[stackSize-1]->median.set((mx[list_size/2]+mx[list_size/2-1])/2, (my[list_size/2]+my[list_size/2-1])/2, (mz[list_size/2]+mz[list_size/2-1])/2);
	}

	stack[stackSize-1]->within = list_size;
	stack[stackSize-1]->p = stack[stackSize-1]->median[stack[stackSize-1]->dim];
	
	while(stackSize > 0){
		while(stack[stackSize-1]->within > 1 && stack[stackSize-1]->l == nullptr){
			short d = stack[stackSize-1]->dim;
			Face** t = new Face*[stack[stackSize-1]->within];
			int s = 0;
			TreeNode* temp = new TreeNode();
			for(int i = 0; i < stack[stackSize-1]->within; i++){
				if(stack[stackSize-1]->contained[i]->median.e[d] < stack[stackSize-1]->median.e[d]){
					t[s] = stack[stackSize-1]->contained[i];
					sortInsertion(s, mx, my, mz, stack[stackSize-1]->contained[i]->median);
					for(int j = 0; j < 3; j++){
						if(t[s]->max[j] > temp->max[j])
							temp->max[j] = t[s]->max[j];
						if(t[s]->min[j] < temp->min[j])
							temp->min[j] = t[s]->min[j];
					}
					s++;
				}
			}
			Face** tt = new Face*[s];
			for(int i = 0; i < s; i++){
				tt[i] = t[i];
			}
			delete[] t;
			if(s > 0){
				temp->dim = d==2?0:d+1;
				if(s % 2 != 0){
					temp->median.set(mx[s/2], my[s/2], mz[s/2]);
				}
				else{
					temp->median.set((mx[s/2]+mx[s/2-1])/2, (my[s/2]+my[s/2-1])/2, (mz[s/2]+mz[s/2-1])/2);
				}
				temp->contained = tt;
				temp->parent = stack[stackSize-1];
				temp->within = s;
				temp->p = temp->median.e[temp->dim];
				stack[stackSize-1]->l = temp;
				if(stackSize < list_size-1){
					stack[stackSize++] = temp;
					maxSize++;
					if(stackSize > 3 && s == stack[stackSize-1]->parent->parent->parent->within){
						stackSize-=3;
						break;
					}
					if(s == 1)
						stack[stackSize-1]->obj = stack[stackSize-1]->contained[0];
				}
				else{
					stackSize--;
					break;
				}
			}
			else{
				break;
			}
			// printf("%d\n", maxSize);
		}
		if(stack[stackSize-1]->within > 1 && stack[stackSize-1]->r == nullptr){
			short d = stack[stackSize-1]->dim;
			Face** t = new Face*[stack[stackSize-1]->within];
			int s = 0;
			// vec3 m;
			TreeNode* temp = new TreeNode();
			// vec3 curMed = stack[stackSize-1]->median;
			for(int i = 0; i < stack[stackSize-1]->within; i++){
				if(stack[stackSize-1]->contained[i]->median.e[d] >= stack[stackSize-1]->median.e[d]){
					t[s] = stack[stackSize-1]->contained[i];
					sortInsertion(s, mx, my, mz, stack[stackSize-1]->contained[i]->median);
					for(int j = 0; j < 3; j++){
						if(t[s]->max[j] > temp->max[j])
							temp->max[j] = t[s]->max[j];
						if(t[s]->min[j] < temp->min[j])
							temp->min[j] = t[s]->min[j];
					}
					s++;
				}
			}
			Face** tt = new Face*[s];
			for(int i = 0; i < s; i++){
				tt[i] = t[i];
			}
			delete[] t;
			if(s > 0){
				temp->dim = d==2?0:d+1;
				if(s % 2 != 0){
					temp->median.set(mx[s/2], my[s/2], mz[s/2]);
				}
				else{
					temp->median.set((mx[s/2]+mx[s/2-1])/2, (my[s/2]+my[s/2-1])/2, (mz[s/2]+mz[s/2-1])/2);
				}
				temp->contained = tt;
				temp->parent = stack[stackSize-1];
				temp->within = s;
				temp->p = temp->median.e[temp->dim];
				stack[stackSize-1]->r = temp;
				if(stackSize < list_size-1){
					stack[stackSize++] = temp;
					maxSize++;
					if(stackSize > 3 && s == stack[stackSize-1]->parent->parent->parent->within){
						stackSize-=3;
						continue;
					}
					if(s == 1)
						stack[stackSize-1]->obj = stack[stackSize-1]->contained[0];
				}
				else stackSize--;
				
			}
			else stackSize--;
		}
		else stackSize--;
		// printf("%d\n", maxSize);
	}

	TriTree* tree = new TriTree;
	tree->head = stack[0];
	tree->numNodes = maxSize;
	return tree;
}

__device__ void TriTree::print(){
	TreeNode* cur = head;//, *t = nullptr;
	printf("numNodes: %d\n", numNodes);
	TreeNode** stack = new TreeNode*[numNodes];
	int stackSize = 0;
	do{
		while(cur != nullptr){
			stack[stackSize++] = cur;
			if(cur->r == nullptr && cur->l == nullptr){
				printf("%p %f %f %f %d\n", cur, cur->median.x(), cur->median.y(), cur->median.z(), cur->within);
				break;
			}
			printf("%p %p %p\n", cur, cur->l, cur->r);
			cur = cur->l;
		}
		while(stackSize > 0 && stack[stackSize-1]->r == nullptr){
			stackSize--;
		}
		if(stackSize > 0){
			cur = stack[stackSize-1]->r;
			stackSize--;
		}
	}	while(stackSize > 0);
	delete[] stack;
}

// hitable* TriTree::copyToDevice(){
// 	return descendCopy(head);
// }

// TreeNode* TriTree::descendCopy(TreeNode* in){
// 	TreeNode *r, *l, *par;
// 	TreeNode* d_in;
// 	Face* d_obj;
// 	TreeNode *oldR = in->r, *oldL = in->l;
// 	if(in->r != nullptr){
// 		r = descendCopy(in->r);
// 	}	else r = nullptr;
// 	if(in->l != nullptr){
// 		l = descendCopy(in->l);
// 	}	else l = nullptr;
// 	hipMalloc((void**)&d_obj, sizeof(Face));
// 	hipMemcpy(Face)
// 	hipMalloc((void**)&d_in, sizeof(TriNode));
// 	// hipMemcpy()
// }