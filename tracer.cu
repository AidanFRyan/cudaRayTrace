#include "hip/hip_runtime.h"
#include "tracer.h"
#include <typeinfo>

vec3::vec3(){
	e[0] = 0;
	e[1] = 0;
	e[2] = 0;
}
vec3::vec3(float e0, float e1, float e2){
	e[0] = e0;
	e[1] = e1;
	e[2] = e2;
}
vec3::vec3(const vec3& v){
	e[0] = v.e[0];
	e[1] = v.e[1];
	e[2] = v.e[2];
}
float vec3::x() const{
	return e[0];
}
float vec3::y() const{
	return e[1];
}
float vec3::z() const{
	return e[2];
}
float vec3::r() const{
	return e[0];
}
float vec3::g() const{
	return e[1];
}
float vec3::b() const{
	return e[2];
}

vec3& vec3::operator=(const vec3& v){
	e[0] = v.e[0];
	e[1] = v.e[1];
	e[2] = v.e[2];
	return *this;
}
const vec3& vec3::operator+() const{
	return *this;
}
vec3 vec3::operator-() const{
	return vec3(-e[0], -e[1], -e[2]);
}
float vec3::operator[](int i) const{
	if(i < 3 && i > 0)
		return e[i];
	else return 0;
}
float& vec3::operator[](int i){
	return e[i];
}

vec3& vec3::operator+=(const vec3 &v2){
	e[0] += v2.e[0];
	e[1] += v2.e[1];
	e[2] += v2.e[2];
	return *this;
}
vec3& vec3::operator-=(const vec3 &v2){
	e[0] -= v2.e[0];
	e[1] -= v2.e[1];
	e[2] -= v2.e[2];
	return *this;
}
vec3& vec3::operator*=(const vec3 &v2){
	e[0] *= v2.e[0];
	e[1] *= v2.e[1];
	e[2] *= v2.e[2];
	return *this;}
vec3& vec3::operator/=(const vec3 &v2){
	e[0] /= v2.e[0];
	e[1] /= v2.e[1];
	e[2] /= v2.e[2];
	return *this;
}
vec3& vec3::operator*=(const float t){
	e[0] *= t;
	e[1] *= t;
	e[2] *= t;
	return *this;
}
vec3& vec3::operator/=(const float t){
	e[0] /= t;
	e[1] /= t;
	e[2] /= t;
	return *this;
}

float vec3::length() const{
	return sqrtf(e[0]*e[0] + e[1]*e[1] + e[2]*e[2]);
}
float vec3::squared_length() const{
	return e[0]*e[0] + e[1]*e[1] + e[2]*e[2];
}
void vec3::make_unit_vector(){
	float k = 1.0f / sqrt(e[0]*e[0] + e[1]*e[1] + e[2] * e[2]);
	e[0] *= k;
	e[0] *= k;
	e[0] *= k;
}

float vec3::dot(const vec3 &v2){
	return e[0]*v2.e[0] + e[1]*v2.e[1] + e[2]*v2.e[2];
}
vec3 vec3::cross(const vec3 &v2){
	return vec3(e[1]*v2.e[2] - e[2]*v2.e[1], (-(e[0]*v2.e[2] - e[2]*v2.e[0])), e[0]*v2.e[1] - e[1]*v2.e[0]);
}

istream& operator>>(istream &is, vec3 &t){
	is>>t.e[0]>>t.e[1]>>t.e[2];
	return is;
}
ostream& operator<<(ostream &os, vec3 &t){
	os<<t.e[0]<<' '<<t.e[1]<<' '<<t.e[2];
	return os;
}
vec3 operator+(const vec3 &v1, const vec3 &v2){
	return vec3(v1.e[0]+v2.e[0], v1.e[1]+v2.e[1], v1.e[2]+v2.e[2]);
}
vec3 operator-(const vec3 &v1, const vec3 &v2){
	return vec3(v1.e[0]-v2.e[0], v1.e[1]-v2.e[1], v1.e[2]-v2.e[2]);
}
vec3 operator*(const vec3 &v1, const vec3 &v2){
	return vec3(v1.e[0]*v2.e[0], v1.e[1]*v2.e[1], v1.e[2]*v2.e[2]);
}
vec3 operator/(const vec3 &v1, const vec3 &v2){
	return vec3(v1.e[0]/v2.e[0], v1.e[1]/v2.e[1], v1.e[2]/v2.e[2]);
}

vec3 operator*(const float t, const vec3 &v){
	return vec3(v.e[0]*t, v.e[1]*t, v.e[2]*t);
}
vec3 operator*(const vec3 &v, const float t){
	return vec3(v.e[0]*t, v.e[1]*t, v.e[2]*t);
}
vec3 operator/(const vec3 v, float t){
	return vec3(v.e[0]/t, v.e[1]/t, v.e[2]/t);
}

float dot(const vec3 &v1, const vec3 &v2){
	return v1.e[0]*v2.e[0] + v1.e[1]*v2.e[1] + v1.e[2]*v2.e[2];
}
vec3 cross(const vec3 &v1, const vec3 &v2){
	return vec3(v1.e[1]*v2.e[2] - v1.e[2]*v2.e[1], (-(v1.e[0]*v2.e[2] - v1.e[2]*v2.e[0])), v1.e[0]*v2.e[1] - v1.e[1]*v2.e[0]);
}

vec3 unit_vector(vec3 v){
	return v/v.length();
}

void vec3::set(float e0, float e1, float e2){
	e[0] = e0;
	e[1] = e1;
	e[2] = e2;
}

ray::ray(){}
ray::ray(const vec3& a, const vec3& b){
	A = a;
	B = b;
}
vec3 ray::origin() const{
	return A;
}
vec3 ray::direction() const{
	return B;
}
vec3 ray::p(float t) const{
	return A + t*B;
}

sphere::sphere(){
	center = vec3(0,0,0);
	radius = 0;
}
sphere::sphere(vec3 cen, float r, material* m){
	center = cen;
	radius = r;
	mat = m;
}
bool sphere::hit(const ray& r, const float& tmin, float& tmax, hit_record& rec) const{
	vec3 oc = r.origin() - center;
	float a = dot(r.direction(), r.direction());
	float b = dot(oc, r.direction());
	float c = dot(oc, oc) - radius*radius;
	float discriminant = b*b-a*c;
	// printf("%f\n", radius);
	if (discriminant > 0){
		rec.mat = mat;
		float temp = (-b - sqrtf(b*b-a*c))/a;
		if (temp < tmax && temp > tmin){
			rec.t = temp;
			rec.p = r.p(rec.t);
			rec.normal = (rec.p - center) / radius;
			return true;
		}
		temp = (-b + sqrtf(b*b-a*c))/a;
		if(temp < tmax && temp > tmin){
			rec.t = temp;
			rec.p = r.p(rec.t);
			rec.normal = (rec.p - center) / radius;
			return true;
		}
	}
	return false;
}

hitable_list::hitable_list(){
	list = 0;
	list_size = 0;
}
hitable_list::hitable_list(hitable **list, int n){
	this->list = list;
	list_size = n;
}
// hitable** hitable_list::listPointer(){
// 	return d_list;
// }
bool hitable_list::hit(const ray& r, const float& tmin, float& tmax, hit_record& rec) const{
	hit_record temp_rec;
	bool anyHits = false;
	float closest = tmax;
	// printf("%p, %d\n", this, this->list_size);
	for(int i = 0; i < list_size; i++){
		// printf("%d %d\n", i, list_size);
		if(list[i]->hit(r, tmin, closest, temp_rec)){
			// printf("%f, %f, %f\n", r.direction().x(), r.direction().y(), r.direction().z());
			anyHits = true;
			closest = temp_rec.t;
			rec = temp_rec;
		}
	}
	return anyHits;
}

//apparently overrides of a parent's virtual functions don't work when the objects are instantiated on the host, instead must be instantiated through a backassward array of pointers and created entirely dynamically on the device
// void hitable_list::copyDevice(){
// 	hitable **h_list = new hitable*[list_size];
// 	// printf("%u\n", sizeof(hitable*));
// 	hipMalloc((void**)&d_list, sizeof(hitable*)*list_size);
// 	hipDeviceSynchronize();
// 	// printf("Done\n");
// 	for(int i = 0; i < list_size; i++){
// 		hitable* temp;
// 		// printf("%s %u\n", typeid(*list[i]).name(), sizeof(*list[i]));
// 		hipMalloc((void**)&temp, sizeof(*list[i]));
// 		hipDeviceSynchronize();
// 		hipMemcpy(temp, list[i], sizeof(*list[i]), hipMemcpyHostToDevice);
// 		hipDeviceSynchronize();
// 		h_list[i] = temp;
// 	}
// 	hipMemcpy(d_list, h_list, sizeof(hitable*)*list_size, hipMemcpyHostToDevice);
// 	hipDeviceSynchronize();
// 	// hitable_list* d_hlist;
// 	hipMalloc((void**)&d_world, sizeof(hitable_list));
// 	hipDeviceSynchronize();
// 	hitable **copy = list;
// 	list = d_list;
// 	// printf("%p %p %p\n", copy, list, h_list);
// 	// for(int i = 0; i < list_size; i++){
// 	// 	printf("copy: %p list: %p hlist: %p\n", copy[i], list[i], h_list[i]);
// 	// }
// 	hipMemcpy(d_world, this, sizeof(hitable_list), hipMemcpyHostToDevice);
// 	hipDeviceSynchronize();
// 	list = copy;
// }
__device__ vec3 random_in_unit_disk(hiprandState* state){
	// hiprandState state;
	// hiprand_init(1234, threadIdx.x+blockDim.x*blockIdx.x, 0, &state);
	vec3 p;
	do{
		p = 2.0f*vec3(hiprand_uniform(state), hiprand_uniform(state), 0) - vec3(1,1,0);
	}while(dot(p,p) >= 1.0f);
	return p;
}

camera::camera(){
	ulc = vec3(-2, 1, -1);
	horizontal = vec3(4, 0, 0);
	vertical = vec3(0,2,0);
	origin = vec3(0,0,0);
}

camera::camera(float vfov, float aspect){
	vfov *= CUDA_PI/180;
	float halfHeight = tanf(vfov/2);
	float halfWidth = aspect*halfHeight;
	ulc = vec3(-halfWidth, halfHeight, -1);
	horizontal = vec3(2*halfWidth, 0, 0);
	vertical = vec3(0,2*halfHeight,0);
	origin = vec3(0,0,0);
}
camera::camera(vec3 o, vec3 lookAt, vec3 vup, float vfov, float aspect){
	// vec3 u, v, w;
	lens_radius=0;
	vfov *= CUDA_PI/180;
	float halfHeight = tanf(vfov/2);
	float halfWidth = aspect*halfHeight;
	origin = o;
	w = unit_vector(o-lookAt);
	u = unit_vector(cross(vup, w));
	v = cross(w, u);
	ulc = vec3(-halfWidth, halfHeight, -1);
	ulc = origin - halfWidth*u + halfHeight*v - w;
	horizontal = 2*halfWidth*u;
	vertical = 2*halfHeight*v;
}
camera::camera(vec3 o, vec3 lookAt, vec3 vup, float vfov, float aspect, float aperture, float focus_dist){
	// vec3 u, v, w;
	lens_radius = aperture/2;
	vfov *= CUDA_PI/180;
	float halfHeight = tanf(vfov/2);
	float halfWidth = aspect*halfHeight;
	origin = o;
	w = unit_vector(o-lookAt);
	u = unit_vector(cross(vup, w));
	v = cross(w, u);
	ulc = vec3(-halfWidth, halfHeight, -1);
	ulc = origin - halfWidth*focus_dist*u + halfHeight*focus_dist*v - focus_dist*w;
	horizontal = 2*halfWidth*u*focus_dist;
	vertical = 2*halfHeight*v*focus_dist;
}
__device__ void camera::get_ray(const float& s, const float& t, ray& r, hiprandState* state){
	vec3 rd = lens_radius * random_in_unit_disk(state);
	vec3 offset = u*rd.x() + v*rd.y();
	r = ray(origin + offset, ulc+s*horizontal-t*vertical-origin-offset);
}

lambertian::lambertian(const vec3& a){
	albedo = a;
	emitter = false;
}

__device__ bool lambertian::scatter(const ray& impacting, const hit_record& rec, vec3& att, ray& scattered, hiprandState* state) const{
	vec3 target = rec.p+rec.normal+random_in_unit_sphere(state);
	scattered = ray(rec.p, target-rec.p);
	att = albedo;
	return true;
}

metal::metal(const vec3& a, const float& f){
	emitter = false;
	albedo = a;
	if(f<1)
		fuzzy = f;
	else
		fuzzy = 1;
}

__device__ vec3 reflect(const vec3& v, const vec3& n){
	return v - 2*dot(v,n)*n;
}

__device__ bool metal::scatter(const ray& impacting, const hit_record& rec, vec3& att, ray& scattered, hiprandState* state) const{
	vec3 reflected = reflect(unit_vector(impacting.direction()), rec.normal);
	if(fuzzy >= 0.01)
		scattered = ray(rec.p, reflected + fuzzy*random_in_unit_sphere(state));
	else
		scattered = ray(rec.p, reflected);
	att = albedo;
	return (dot(scattered.direction(), rec.normal) > 0);
}

dielectric::dielectric(const float& i){
	ior = i;
	emitter = false;
}

__device__  bool dielectric::scatter(const ray& impacting, const hit_record& rec, vec3& att, ray& scattered, hiprandState* state) const{
	// int index = threadIdx.x + blockDim.x*blockIdx.x;
	// if (index == 0)
	// 		printf("scattering %f, %f, %f\n", impacting.direction().e[0], impacting.direction().e[1], impacting.direction().e[2]);//, outward_normal.e[0], outward_normal.e[1], outward_normal.e[2]);
	vec3 outward_normal;
	vec3 reflected = reflect(impacting.direction(), rec.normal);
	float ni_nt;
	att = vec3(1.0f, 1.0f, 1.0f);
	vec3 refracted;
	float reflect_prob;
	float cosine;
	float dotted = dot(impacting.direction(), rec.normal);
	if(dotted>0){//if normal and ray are facing same direction
		outward_normal = -rec.normal;
		// if (index == 0)
		// 	printf("%d %f, %f, %f; %f, %f, %f\n", index, rec.normal.e[0], rec.normal.e[1], rec.normal.e[2], outward_normal.e[0], outward_normal.e[1], outward_normal.e[2]);
		ni_nt = ior;
		cosine = dotted/impacting.direction().length();
		cosine = sqrtf(1-ior*ior*(1-cosine*cosine));
	}
	else{
		outward_normal = rec.normal;
		ni_nt = 1.0f/ior;
		cosine = -dotted/impacting.direction().length();
		// cosine = sqrtf(1-ior*ior*(1-cosine*cosine));
	}
	if(refract(impacting.direction(), outward_normal, ni_nt, refracted)){
		
		// if (index == 0)
		// 	printf("scattering %f, %f, %f; %f, %f, %f\n", impacting.direction().e[0], impacting.direction().e[1], impacting.direction().e[2], outward_normal.e[0], outward_normal.e[1], outward_normal.e[2]);
		// printf("%d %p\n", threadIdx.x + blockDim.x*blockIdx.x, &outward_normal);
		reflect_prob = schlick(cosine, ior);
	}
	else{
		reflect_prob = 1;
	}
	if(hiprand_uniform(state) < reflect_prob){
		scattered = ray(rec.p, reflected);
	}
	else{
		scattered = ray(rec.p, refracted);
		// printf("refracted\n");
	}
	return true;
}

__device__ bool refract(const vec3& v, const vec3& n, float ni_nt, vec3& refracted){
	vec3 uv = unit_vector(v);
	// vec3 un = unit_vector(n);
	float dt = dot(uv, n);
	// pr  intf("%f\n", dt);
	float discriminant = 1.0f-ni_nt*ni_nt*(1.0f-dt*dt);
	if(discriminant > 0){
		refracted = ni_nt*(uv-n*dt) - n*sqrtf(discriminant);
		// int index = threadIdx.x + blockDim.x*blockIdx.x;
		// if (index == 0)
		// 	printf("refracted %f, %f, %f; %f, %f, %f\n", v.e[0], v.e[1], v.e[2], refracted.e[0], refracted.e[1], refracted.e[2]);
			// printf("%d %p\n", index, &n);
		return true;
	}
	else return false;
}

__device__ float dielectric::schlick(const float& cosine, const float& indor) const{
	float r0 = (1-indor)/(1+indor);
	r0 = r0*r0;
	return r0 + (1-r0)*pow((1-cosine), 5);
}

__device__ vec3 random_in_unit_sphere(hiprandState* state){
	// hiprandState state;
	// printf("Finding rand\n");
	vec3 p;
	do {
		p = 2*vec3(hiprand_uniform(state),hiprand_uniform(state),hiprand_uniform(state)) - vec3(1,1,1);
	} while(p.squared_length() >= 1);
	return p;
}

// __device__ bool refract(const vec3&  v, const vec3& n, const float& ni_over_nt, vec3& refracted){
// 	vec3 uv = unit_vector(v);
// 	float dt = dot(uv, n);
// 	float discriminant = 1.0-ni_over_nt*ni_over_nt*(1-dt*dt);
// 	if(discriminant > 0){
// 		refracted = ni_over_nt*(uv - n*dt) - n*sqrtf(discriminant);
// 		return true;
// 	}
// 	else return false;
// }

// __device__ bool dielectric::scatter(const ray& r_in, const hit_record& rec, vec3& attenuation, ray& scattered, hiprandState* state) const{
// 	vec3 outward_normal;
// 	vec3 reflected = reflect(r_in.direction(), rec.normal);
// 	float ni_over_nt;
// 	attenuation = vec3(1.0f, 1.0f, 1.0f);
// 	vec3 refracted;
// 	if(dot(r_in.direction(), rec.normal)>0){
// 		outward_normal = -rec.normal;
// 		ni_over_nt = ref_idx;
// 	}
// 	else{
// 		outward_normal = rec.normal;
// 		ni_over_nt = 1.0f/ref_idx;
// 	}
// 	if(refract(r_in.direction(), outward_normal, ni_over_nt, refracted)){
// 		scattered = ray(rec.p, refracted);
// 	}
// 	else{
// 		scattered = ray(rec.p, reflected);
// 		return false;
// 	}
// 	return true;
// }

__device__ light::light(vec3 att){
	attenuation = att;
	emitter = true;
}

__device__ bool light::scatter(const ray& impacting, const hit_record& rec, vec3& att, ray& scattered, hiprandState* state) const{
	att = attenuation;
	scattered = impacting;
	return true;
}