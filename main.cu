#include "hip/hip_runtime.h"
#include "tracer.h"
#include <OpenEXR/ImfNamespace.h>
#include <OpenEXR/ImfOutputFile.h>
#include <OpenEXR/ImfChannelList.h>

using namespace OPENEXR_IMF_NAMESPACE;

__global__ void worldGenerator(hitable** list, hitable_list** world, int wSize){
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	if(i==0){
		// hitable* list[2];
		list[0] = new sphere(vec3(0,0,-1), 0.5, new lambertian(vec3(0.8f, 0.3f, 0.3f)));
		list[1] = new sphere(vec3(0,-100.5, -1), 100, new lambertian(vec3(0.8f, 0.8f, 0.0f)));
		list[2] = new sphere(vec3(1, 0, -1), 0.5, new metal(vec3(0.8f, 0.6f, 0.2f), 0.0f));
		// list[3] = new sphere(vec3(-1, 0, -1), 0.5, new metal(vec3(0.8f, 0.8f, 0.8f), 1.0f))
		list[5] = new sphere(vec3(-1, 0, -1), 0.5f, new dielectric(1.5f));
		list[3] = new sphere(vec3(2, 1, 0), 0.5f, new light(vec3(2, 2, 2)));
		list[4] = new sphere(vec3(-1, 1, -2), 0.5f, new light(vec3(4, 2, 2)));
		// list[4] = new sphere(vec3(0,1,-1), 0.5f, new metal(vec3(0.8f, 0.8f, 0.9f), 0));
		*world = new hitable_list(list, wSize);
	}
}

__global__ void initRand(int n, int cluster, int aa, hiprandState* state){
	int index = threadIdx.x+blockDim.x*blockIdx.x;
	int pixelNum = index*cluster;
	while(pixelNum < n){
		
		for(int i = 0; i < cluster && (pixelNum+i) < n; i++){
			hiprand_init(pixelNum+i, pixelNum+i, 0, &state[pixelNum+i]);
		}
		pixelNum += blockDim.x*gridDim.x;
	}
}

__device__ vec3 color(const ray& r, hitable_list* world, hiprandState* state){
	
	float max = FLT_MAX;
	ray curRay = r;
	vec3 curLight = vec3(1,1,1);
	for(int i = 0; i < 20; i++){
		hit_record rec;
		if(world->hit(curRay, 0.00001, max, rec)){
			ray scattered;
			vec3 attenuation;
			if(rec.mat->emitter && rec.mat->scatter(r, rec, attenuation, scattered, state)){
				curLight *= attenuation;
				return curLight;
			}
			else if(rec.mat->scatter(r, rec, attenuation, scattered, state)){
				curLight *= attenuation;
				curRay = scattered;
			}
			else{
				return vec3(0,0,0);
			}

		}
		else{
			return vec3(0,0,0);
			vec3 unit_direction = unit_vector(curRay.direction());
			float t = 0.5f*(unit_direction.y()+1.0f);
			vec3 c = (1.0f-t)*vec3(1, 1, 1) + t*vec3(0.5f, 0.7f, 1);
			return curLight * c;
		}
	}
	return vec3(0.0f, 0.0f, 0.0f);
}

__global__ void imageGenerator(int x, int y, int cluster, camera cam, int aa, hitable_list** world, vec3* img, hiprandState* state){
	
	int index = threadIdx.x + blockDim.x*blockIdx.x;
	int pixelNum = index*cluster;
	
	while(pixelNum < x*y){

		for(int i = 0; i < cluster && (pixelNum+i) < x*y; i++){
			float pixX = (pixelNum+i)%x, pixY = (pixelNum+i)/x;

			
			
			vec3 col;
			for(int j = 0; j < aa; j++){
				float u, v;
				u = (pixX+hiprand_uniform(&state[pixelNum+i])) / x;
				v = (pixY+hiprand_uniform(&state[pixelNum+i])) / y;
				ray r;
				cam.get_ray(u, v, r, &state[pixelNum+i]);
				col += color(r, *world, &state[pixelNum+i]);
			}
			col /= aa;
			img[pixelNum+i].set(col[0], col[1], col[2]);
		}
		pixelNum += blockDim.x*gridDim.x;
	}
}

__global__ void averageImgs(vec3* fin, vec3** img1, int count, int x, int y, float* r, float* g, float* b, float* a){
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	int pixelNum = index;
	while(pixelNum < x*y){
		for(int i = 0; i < count; i++){
			fin[pixelNum] += img1[i][pixelNum];
		}
		fin[pixelNum]/=count;
		r[pixelNum] = fin[pixelNum].r();
		g[pixelNum] = fin[pixelNum].g();
		b[pixelNum] = fin[pixelNum].b();
		a[pixelNum] = 1.0f;
		pixelNum += gridDim.x*blockDim.x;
		
	}
}

int main(){
	hiprandState** state;
	hitable *** list;
	hitable_list ***world;// = new hitable_list(list, 2);
	int worldSize = 6;
	int count;
	hipGetDeviceCount(&count);

	state = new hiprandState*[count];
	list = new hitable**[count];
	world = new hitable_list**[count];
	int x = 1920;
	int y = 1080;
	int aaSamples = 1024;
	vec3 **imgBuf, **d_img;//, origin(0,0,0), ulc(-2,1,-1), hor(4,0,0), vert(0,2,0);
	d_img = new vec3*[count];
	imgBuf = new vec3*[count];
	vec3 lookFrom(-3,3,2);
	vec3 lookAt(0,0,-1);
	float dist = (lookFrom-lookAt).length();
	float ap = 0.0f;
	camera cam(lookFrom, lookAt, vec3(0, 1, 0), 20, float(x)/float(y), ap, dist);
	// hitable *list[2];
	for(int i = 0; i < count; i++){
		
		hipSetDevice(i);
		
		hipMalloc((void**)&state[i], x*y*sizeof(hiprandState));
		hipMalloc((void**)&world[i], sizeof(hitable_list*));
		hipMalloc((void**)&list[i], worldSize*sizeof(hitable*));
	}
	hipDeviceSynchronize();
	for(int i = 0; i < count; i++){
		hipSetDevice(i);
		initRand<<<4,512>>>(x*y, 1, aaSamples/count, state[i]);
	}
	hipDeviceSynchronize();
	for(int i = 0; i < count; i++){
		hipSetDevice(i);
		
		worldGenerator<<<1,1>>>(list[i], world[i], worldSize);
		hipMalloc((void**)&d_img[i], sizeof(vec3)*x*y);
	}
	hipDeviceSynchronize();
	for(int i = 0; i < count; i++){
		hipSetDevice(i);
		imageGenerator<<<4, 512>>>(x, y, 1, cam, aaSamples/count, world[i], d_img[i], state[i]);
		imgBuf[i] = new vec3[x*y];	
	}

	for(int i = 0; i < count; i++){
		hipSetDevice(i);
		hipDeviceSynchronize();
		
		hipMemcpy(imgBuf[i], d_img[i], sizeof(vec3)*x*y, hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		hipFree(state[i]);
		hipFree(world[i]);
		hipFree(list[i]);
		hipFree(d_img[i]);
	}
	delete[] state;
	delete[] world;
	delete[] list;
	delete[] d_img;

	hipSetDevice(count-1);
	hipDeviceSynchronize();
	
	vec3** d_imgs, **imgs;
	vec3* finImg, *img;

	imgs = new vec3*[count];
	hipMalloc((void**)&d_imgs, count*sizeof(vec3*));
	hipMalloc((void**)&finImg, sizeof(vec3)*x*y);
	img = new vec3[x*y];

	for(int i = 0; i < count; i++){
		hipMalloc((void**)&imgs[i], x*y*sizeof(vec3));
	}
	hipDeviceSynchronize();
	for(int i = 0; i < count; i++){
		hipMemcpy(imgs[i], imgBuf[i], sizeof(vec3)*x*y, hipMemcpyHostToDevice);
	}
	hipMemcpy(d_imgs, imgs, count*sizeof(vec3*), hipMemcpyHostToDevice);
	
	float *d_r, *d_g, *d_b, *d_a;
	float *r, *g, *b, *a;
	hipMalloc((void**)&d_r, sizeof(float)*x*y);
	hipMalloc((void**)&d_g, sizeof(float)*x*y);
	hipMalloc((void**)&d_b, sizeof(float)*x*y);
	hipMalloc((void**)&d_a, sizeof(float)*x*y);
	hipDeviceSynchronize();

	averageImgs<<<4, 512>>>(finImg, d_imgs, count, x, y, d_r, d_g, d_b, d_a);
	r = new float[x*y];
	g = new float[x*y];
	b = new float[x*y];
	a = new float[x*y];
	hipDeviceSynchronize();

	hipMemcpy(img, finImg, sizeof(vec3)*x*y, hipMemcpyDeviceToHost);
	hipMemcpy(r, d_r, sizeof(float)*x*y, hipMemcpyDeviceToHost);
	hipMemcpy(g, d_g, sizeof(float)*x*y, hipMemcpyDeviceToHost);
	hipMemcpy(b, d_b, sizeof(float)*x*y, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	hipFree(d_r);
	hipFree(d_g);
	hipFree(d_b);
	hipFree(d_a);
	hipFree(d_imgs);
	hipFree(finImg);
	delete[] imgs;

	Header header(x, y);
	header.channels().insert("R", Channel(FLOAT));
	header.channels().insert("G", Channel(FLOAT));
	header.channels().insert("B", Channel(FLOAT));
	header.channels().insert("A", Channel(FLOAT));

	OutputFile file("out.exr", header);

	FrameBuffer frameBuffer;
	frameBuffer.insert("R", Slice(FLOAT, (char*)r, sizeof(*r)*1, sizeof(*r)*x));
	frameBuffer.insert("G", Slice(FLOAT, (char*)g, sizeof(*g)*1, sizeof(*g)*x));
	frameBuffer.insert("B", Slice(FLOAT, (char*)b, sizeof(*b)*1, sizeof(*b)*x));
	frameBuffer.insert("A", Slice(FLOAT, (char*)a, sizeof(*a)*1, sizeof(*a)*x));
	file.setFrameBuffer(frameBuffer);
	file.writePixels(y);
	
	// cout<<"P3\n"<<x<<' '<<y<<"\n255\n";
	// for(int i = 0; i < x*y; i++){
	// 	cout<<img[i].r()<<' '<<img[i].g()<<' '<<img[i].b()<<'\n';
	// }
	// delete[] imgBuf;

	delete[] r;
	delete[] g;
	delete[] b;
	delete[] a;
	delete[] img;
	return 0;
}