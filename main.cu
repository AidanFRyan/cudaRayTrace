#include "hip/hip_runtime.h"
#include "tracer.h"
// #include "objRead.h"
#include <OpenEXR/ImfNamespace.h>
#include <OpenEXR/ImfOutputFile.h>
#include <OpenEXR/ImfChannelList.h>



using namespace OPENEXR_IMF_NAMESPACE;

__global__ void worldGenerator(hitable** list, hitable_list** world, int wSize, OBJ** objs, int numOBJs, int cluster){
	int index = threadIdx.x + blockDim.x*blockIdx.x;
	if(index==0){
		// hitable* list[2];
		// list[0] = new sphere(vec3(0,0,-1), 0.5, new lambertian(vec3(0.8f, 0.3f, 0.3f)));
		// list[1] = new sphere(vec3(0,-100.5, -1), 100, new lambertian(vec3(0.8f, 0.8f, 0.0f)));
		// list[2] = new sphere(vec3(1, 0, -1), 0.5, new metal(vec3(0.8f, 0.6f, 0.2f), 0.0f));
		// // list[3] = new sphere(vec3(-1, 0, -1), 0.5, new metal(vec3(0.8f, 0.8f, 0.8f), 1.0f))
		// list[5] = new sphere(vec3(-1, 0, -1), 0.5f, new dielectric(1.5f));
		// list[3] = new sphere(vec3(2, 1, 0), 0.5f, new light(vec3(2, 2, 2)));
		// list[4] = new sphere(vec3(-1, 1, -2), 0.5f, new light(vec3(4, 2, 2)));
		// list[4] = new sphere(vec3(0,1,-1), 0.5f, new metal(vec3(0.8f, 0.8f, 0.9f), 0));
		// int totalFaces = 0;
		// for(int j = 0; j < numOBJs; j++){
		// 	totalFaces += objs[j]->numFaces;
		// }
		// hitable** worldFaces = new hitable*[totalFaces];
		// int zz = 0;
		// for(int j = 0; j < numOBJs; j++){
		// 	for(int z = 0; z < objs[j]->numFaces; z++){
		// 		worldFaces[zz] = new Face(objs[j]->object[z]);
		// 	}
		// }
		// printf("Trying to create hitable_list\n");
		// printf("numObjs: %d\n", numOBJs);
		
		*world = new hitable_list(objs, numOBJs);
	}
	__syncthreads();
	int curIndex = index*cluster;
	
	while(curIndex < (*world)->list_size){
		for(int i = 0; i < cluster && (curIndex+i) < (*world)->list_size; i++){
			// int z = 0;
			// for(int i = 0; i < n; i++){
			// 	for(int j = 0; j < in[i]->numFaces; j++){
			// 		// printf("j: %d\n", j);
			// 		list[z] = new Face(in[i]->object[j], new light(vec3(4, 2, 2)));
			// 		// *list[z] = in[z];
			// 		z++;
			// 		if(z%10000 == 0)
			// 			printf("%d\n", z);
			// 	}
			// }
			int totalFaces = 0, offset = 0;
			for(int j = 0; j < numOBJs; j++){
				totalFaces += objs[j]->numFaces;
				if(curIndex+i < totalFaces){
					for(int z = 0; z < j; z++){
						offset += objs[j]->numFaces;
					}
					(*world)->list[curIndex+i] = new Face(objs[j]->object[curIndex+i-offset], new light(vec3(4, 2, 2)));
					// printf("%d %p\n", curIndex+i, (*world)->list[curIndex+i]);
				}				
			}
		}
		curIndex += gridDim.x*blockDim.x;
	}
}

__global__ void initRand(int n, int cluster, int aa, hiprandState* state){
	int index = threadIdx.x+blockDim.x*blockIdx.x;
	int pixelNum = index*cluster;
	while(pixelNum < n){
		
		for(int i = 0; i < cluster && (pixelNum+i) < n; i++){
			hiprand_init(pixelNum+i, pixelNum+i, 0, &state[pixelNum+i]);
		}
		pixelNum += blockDim.x*gridDim.x;
	}
}

__device__ vec3 color(const ray& r, hitable_list* world, hiprandState* state){
	
	float max = FLT_MAX;
	ray curRay = r;
	vec3 curLight = vec3(1,1,1);
	for(int i = 0; i < 10; i++){
		hit_record rec;
		if(world->hit(curRay, 0.00001f, max, rec)){
			ray scattered;
			vec3 attenuation;
			if(rec.mat->emitter && rec.mat->scatter(r, rec, attenuation, scattered, state)){
				curLight *= attenuation;
				return curLight;
			}
			else if(rec.mat->scatter(r, rec, attenuation, scattered, state)){
				curLight *= attenuation;
				curRay = scattered;
			}
			else{
				return vec3(0,0,0);
			}

		}
		else{
			return vec3(0,0,0);
			vec3 unit_direction = unit_vector(curRay.direction());
			float t = 0.5f*(unit_direction.y()+1.0f);
			vec3 c = (1.0f-t)*vec3(1, 1, 1) + t*vec3(0.5f, 0.7f, 1);
			return curLight * c;
		}
	}
	return vec3(0.0f, 0.0f, 0.0f);
}

__global__ void imageGenerator(int x, int y, int cluster, camera cam, int aa, hitable_list** world, vec3* img, hiprandState* state){
	
	int index = threadIdx.x + blockDim.x*blockIdx.x;
	int pixelNum = index*cluster;
	
	while(pixelNum < x*y){

		for(int i = 0; i < cluster && (pixelNum+i) < x*y; i++){
			float pixX = (pixelNum+i)%x, pixY = (pixelNum+i)/x;

			
			
			vec3 col;
			for(int j = 0; j < aa; j++){
				float u, v;
				u = (pixX+hiprand_uniform(&state[pixelNum+i])) / x;
				v = (pixY+hiprand_uniform(&state[pixelNum+i])) / y;
				ray r;
				cam.get_ray(u, v, r, &state[pixelNum+i]);
				col += color(r, *world, &state[pixelNum+i]);
			}
			col /= aa;
			img[pixelNum+i].set(col[0], col[1], col[2]);
		}
		// if(index == 0)
		// 	printf("%f%% finished\n", (float(pixelNum)/(x*y))*100);
		pixelNum += blockDim.x*gridDim.x;
	}
}

__global__ void averageImgs(vec3* fin, vec3** img1, int count, int x, int y, float* r, float* g, float* b, float* a){
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	int pixelNum = index;
	while(pixelNum < x*y){
		for(int i = 0; i < count; i++){
			fin[pixelNum] += img1[i][pixelNum];
		}
		fin[pixelNum]/=count;
		r[pixelNum] = fin[pixelNum].r();
		g[pixelNum] = fin[pixelNum].g();
		b[pixelNum] = fin[pixelNum].b();
		a[pixelNum] = 1.0f;
		pixelNum += gridDim.x*blockDim.x;
		
	}
}

__global__ void clearWorld(hitable_list ** world, int cluster){
	int index = threadIdx.x + blockDim.x*blockIdx.x;
	int curIndex = index*cluster;
	while(curIndex < (*world)->list_size){
		for(int i = 0; i < cluster && (curIndex+i) < (*world)->list_size; i++){
			delete (*world)->list[curIndex+i];
		}
		curIndex+=gridDim.x*blockDim.x;
	}
	__syncthreads();
	if(index == 0)
	delete[] (*world)->list;
}

int main(int argc, char* argv[]){
	// printf("%d\n", sizeof(Face));

	// printf("%d\n", argc);
	size_t totalSize = 0, *curSize = new size_t;
	int numOBJs = argc-1;
	OBJ ***d_objs, **objs = new OBJ*[numOBJs], ***h_d_objs;// = new OBJ*[numOBJs];
	for(int i = 0; i < numOBJs; i++){
		objs[i] = new OBJ(argv[i+1]);
		totalSize += objs[i]->numFaces*sizeof(Face) + objs[i]->numP*sizeof(vec3) + objs[i]->numT*sizeof(vec3) + objs[i]->numN*sizeof(vec3);
	}
	printf("Read .objs\n");
	hiprandState** state;
	hitable *** list;
	hitable_list ***world;// = new hitable_list(list, 2);
	int worldSize = 6;
	int count, firstDevice = 0;
	gpuErrchk(hipGetDeviceCount(&count));
	// printf("numDevices: %d\n", count);
	state = new hiprandState*[count];
	list = new hitable**[count];
	world = new hitable_list**[count];

	int x = 720;
	int y = 480;
	int aaSamples = 2;

	vec3 **imgBuf, **d_img;//, origin(0,0,0), ulc(-2,1,-1), hor(4,0,0), vert(0,2,0);
	d_img = new vec3*[count];
	imgBuf = new vec3*[count];
	d_objs = new OBJ**[count];
	h_d_objs = new OBJ**[count];
	vec3 lookFrom(-10, -10, 10);
	vec3 lookAt(0,0,0);
	float dist = (lookFrom-lookAt).length();
	float ap = 0.0f;
	camera cam(lookFrom, lookAt, vec3(0, 1, 0), 40, float(x)/float(y), ap, dist);
	// hitable *list[2];
	printf("Beginning World Allocation, allocating %d bytes\n", totalSize);
	totalSize*=4;
	for(int i = firstDevice; i < count; i++){
		// printf("%d\n", i);
		
		gpuErrchk(hipSetDevice(i));
		gpuErrchk(hipDeviceSetLimit(hipLimitMallocHeapSize, totalSize));
		hipDeviceSynchronize();
		hipDeviceGetLimit(curSize, hipLimitMallocHeapSize);
		gpuErrchk(hipMalloc((void**)&state[i], x*y*sizeof(hiprandState)));
		gpuErrchk(hipMalloc((void**)&world[i], sizeof(hitable_list*)));
		gpuErrchk(hipMalloc((void**)&list[i], worldSize*sizeof(hitable*)));
		h_d_objs[i] = new OBJ*[numOBJs];
	}
	hipDeviceSynchronize();
	printf("Beginning Rand Generation, %d bytes allocated\n", totalSize);
	for(int i = firstDevice; i < count; i++){
		// printf("%d\n", i);
		gpuErrchk(hipSetDevice(i));
		initRand<<<4,512>>>(x*y, 1, aaSamples/count, state[i]);
	}
	gpuErrchk(hipDeviceSynchronize());
	printf("Beginning Copy of Faces to Device\n");
	for(int i = firstDevice; i < count; i++){
		gpuErrchk(hipSetDevice(i));
		for(int j = 0; j < numOBJs; j++){
			// printf("%d %d\n", i, j);
			h_d_objs[i][j] = objs[j]->copyToDevice();
		}
		gpuErrchk(hipMalloc((void**)&d_objs[i], sizeof(OBJ*)*numOBJs));
		gpuErrchk(hipMemcpy(d_objs[i], h_d_objs[i], sizeof(OBJ*)*numOBJs, hipMemcpyHostToDevice));
		gpuErrchk(hipDeviceSynchronize());
	}
	hipDeviceSynchronize();
	printf("worldGenerator Beginning\n");
	for(int i = firstDevice; i < count; i++){
		hipSetDevice(i);
		
		worldGenerator<<<1,1024>>>(list[i], world[i], worldSize, d_objs[i], numOBJs, 1);
		hipMalloc((void**)&d_img[i], sizeof(vec3)*x*y);
	}
	hipDeviceSynchronize();
	printf("Beginning Render\n");
	for(int i = firstDevice; i < count; i++){
		hipSetDevice(i);
		imageGenerator<<<4, 512>>>(x, y, 1, cam, aaSamples/count, world[i], d_img[i], state[i]);
		imgBuf[i] = new vec3[x*y];	
	}
	hipDeviceSynchronize();
	printf("Done With Rendering, Copying to Disk/Cleaning\n");
	for(int i = firstDevice; i < count; i++){
		hipSetDevice(i);
		
		
		hipMemcpy(imgBuf[i], d_img[i], sizeof(vec3)*x*y, hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		hipFree(state[i]);
		hipFree(world[i]);
		hipFree(list[i]);
		hipFree(d_img[i]);
		clearWorld<<<1, 1024>>>(world[i], 1);
	}
	printf("Done Cleaning, Merging from devices\n");
	delete[] state;
	delete[] world;
	delete[] list;
	delete[] d_img;

	hipSetDevice(count-1);
	hipDeviceSynchronize();
	
	vec3** d_imgs, **imgs;
	vec3* finImg, *img;

	imgs = new vec3*[count];
	hipMalloc((void**)&d_imgs, count*sizeof(vec3*));
	hipMalloc((void**)&finImg, sizeof(vec3)*x*y);
	img = new vec3[x*y];

	for(int i = 0; i < count; i++){
		hipMalloc((void**)&imgs[i], x*y*sizeof(vec3));
	}
	hipDeviceSynchronize();
	for(int i = 0; i < count; i++){
		hipMemcpy(imgs[i], imgBuf[i], sizeof(vec3)*x*y, hipMemcpyHostToDevice);
	}
	hipMemcpy(d_imgs, imgs, count*sizeof(vec3*), hipMemcpyHostToDevice);
	
	float *d_r, *d_g, *d_b, *d_a;
	float *r, *g, *b, *a;
	hipMalloc((void**)&d_r, sizeof(float)*x*y);
	hipMalloc((void**)&d_g, sizeof(float)*x*y);
	hipMalloc((void**)&d_b, sizeof(float)*x*y);
	hipMalloc((void**)&d_a, sizeof(float)*x*y);
	hipDeviceSynchronize();

	averageImgs<<<4, 512>>>(finImg, d_imgs, count, x, y, d_r, d_g, d_b, d_a);
	r = new float[x*y];
	g = new float[x*y];
	b = new float[x*y];
	a = new float[x*y];
	hipDeviceSynchronize();

	hipMemcpy(img, finImg, sizeof(vec3)*x*y, hipMemcpyDeviceToHost);
	hipMemcpy(r, d_r, sizeof(float)*x*y, hipMemcpyDeviceToHost);
	hipMemcpy(g, d_g, sizeof(float)*x*y, hipMemcpyDeviceToHost);
	hipMemcpy(b, d_b, sizeof(float)*x*y, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	hipFree(d_r);
	hipFree(d_g);
	hipFree(d_b);
	hipFree(d_a);
	hipFree(d_imgs);
	hipFree(finImg);
	delete[] imgs;

	Header header(x, y);
	header.channels().insert("R", Channel(FLOAT));
	header.channels().insert("G", Channel(FLOAT));
	header.channels().insert("B", Channel(FLOAT));
	header.channels().insert("A", Channel(FLOAT));

	OutputFile file("out.exr", header);

	FrameBuffer frameBuffer;
	frameBuffer.insert("R", Slice(FLOAT, (char*)r, sizeof(*r)*1, sizeof(*r)*x));
	frameBuffer.insert("G", Slice(FLOAT, (char*)g, sizeof(*g)*1, sizeof(*g)*x));
	frameBuffer.insert("B", Slice(FLOAT, (char*)b, sizeof(*b)*1, sizeof(*b)*x));
	frameBuffer.insert("A", Slice(FLOAT, (char*)a, sizeof(*a)*1, sizeof(*a)*x));
	file.setFrameBuffer(frameBuffer);
	file.writePixels(y);
	
	// cout<<"P3\n"<<x<<' '<<y<<"\n255\n";
	// for(int i = 0; i < x*y; i++){
	// 	cout<<img[i].r()<<' '<<img[i].g()<<' '<<img[i].b()<<'\n';
	// }
	// delete[] imgBuf;

	delete[] r;
	delete[] g;
	delete[] b;
	delete[] a;
	delete[] img;
	return 0;
}