#include "hip/hip_runtime.h"
//Aidan Ryan, 2019

#include "tracer.h"
// #include "objRead.h"
#include "kerns.hu"

#include <OpenEXR/ImfNamespace.h>
#include <OpenEXR/ImfOutputFile.h>
#include <OpenEXR/ImfChannelList.h>
#include <hip/hip_cooperative_groups.h>


using namespace OPENEXR_IMF_NAMESPACE;
using namespace cooperative_groups;

int main(int argc, char* argv[]){
	size_t totalSize = 0, *curSize = new size_t;
	int numOBJs = argc-1;
	OBJ ***d_objs, **objs = new OBJ*[numOBJs], ***h_d_objs;// = new OBJ*[numOBJs];
	
	printf("Read .objs\n");
	hiprandState** state;
	hitable *** list;
	hitable_list ***world;// = new hitable_list(list, 2);
	int worldSize = 10;
	int count, firstDevice = 0;
	gpuErrchk(hipGetDeviceCount(&count));
	// printf("numDevices: %d\n", count);
	state = new hiprandState*[count];
	list = new hitable**[count];
	world = new hitable_list**[count];

	int numBlocks = 1, numThreads = 512;
	
	int x = 200;
	int y = 100;

	// x = 1000;
	// y = 500;
	int aaSamples = 32;

	vec3 **imgBuf, **d_img;//, origin(0,0,0), ulc(-2,1,-1), hor(4,0,0), vert(0,2,0);
	d_img = new vec3*[count];
	imgBuf = new vec3*[count];
	d_objs = new OBJ**[count];
	h_d_objs = new OBJ**[count];
	vec3 lookFrom(5, 2, 5);
	vec3 lookAt(0,0,0);
	float dist = (lookFrom-lookAt).length();
	float ap = 0.0f;
	camera cam(lookFrom, lookAt, vec3(0, 1, 0), 60, float(x)/float(y), ap, dist);
	// hitable *list[2];
	int numObjs = worldSize;

	TreeNode**** h_d_nodes = new TreeNode***[count], ****d_nodes = new TreeNode***[count];

	for(int i = 0; i < numOBJs; i++){
		objs[i] = new OBJ(argv[i+1]);
		totalSize += objs[i]->numFaces*sizeof(TreeNode) + objs[i]->numFaces*objs[i]->numFaces*sizeof(Face*) + objs[i]->numFaces*sizeof(TreeNode*) + numBlocks*numThreads*objs[i]->numFaces*sizeof(TreeNode*) + numBlocks*numThreads*objs[i]->numFaces*sizeof(bool) + objs[i]->numP*sizeof(vec3) + objs[i]->numT*sizeof(vec3) + objs[i]->numN*sizeof(vec3) + objs[i]->numFaces*sizeof(hit_record);//+objs[i]->numFaces*sizeof(bool)+objs[i]->numFaces*sizeof(hit_record)+objs[i]->numFaces*sizeof(float);// + x*y*(objs[i]->numFaces*(sizeof(bool)+sizeof(hit_record)+sizeof(float)));
		numObjs += objs[i]->numFaces;
	}
	printf("Beginning World Allocation, allocating %u bytes\n", totalSize);
	for(int i = 0; i < count; i++){
		gpuErrchk(hipSetDevice(i));
		gpuErrchk(hipDeviceSetLimit(hipLimitMallocHeapSize, totalSize));
		hipDeviceSynchronize();
		hipDeviceGetLimit(curSize, hipLimitMallocHeapSize);
		gpuErrchk(hipMalloc((void**)&state[i], x*y*sizeof(hiprandState)));
		gpuErrchk(hipMalloc((void**)&world[i], sizeof(hitable_list*)));
		gpuErrchk(hipMalloc((void**)&list[i], worldSize*sizeof(hitable*)));
		h_d_objs[i] = new OBJ*[numOBJs];
	}
	hipDeviceSynchronize();
	printf("Beginning Rand Generation, %u bytes allocated\n", totalSize);
	for(int i = 0; i < count; i++){
		// printf("%d\n", i);
		gpuErrchk(hipSetDevice(i));
		initRand<<<4,512>>>(x*y, 1, aaSamples/count, state[i]);
	}
	gpuErrchk(hipDeviceSynchronize());
	printf("Beginning Copy of Faces to Device\n");
	for(int i = 0; i < count; i++){
		hipSetDevice(i);
		// hipMalloc((void**)&h_d_nodes[i], sizeof(TreeNode**)*numOBJs);
		h_d_nodes[i] = new TreeNode**[numOBJs];
		for(int j = 0; j < numOBJs; j++){
			h_d_objs[i][j] = objs[j]->copyToDevice();
			
			hipMalloc((void**)&h_d_nodes[i][j], sizeof(TreeNode*)*(3*(objs[j]->numFaces*2+1)));
		}

		hipMalloc((void**)&d_nodes[i], sizeof(TreeNode**)*numOBJs);
		hipMemcpy(d_nodes[i], h_d_nodes[i], sizeof(TreeNode**)*numOBJs, hipMemcpyHostToDevice);

		hipMalloc((void**)&d_objs[i], sizeof(OBJ*)*numOBJs);
		hipMemcpy(d_objs[i], h_d_objs[i], sizeof(OBJ*)*numOBJs, hipMemcpyHostToDevice);

		hipDeviceSynchronize();
	}	
	printf("worldGenerator Beginning\n");
	for(int i = 0; i < count; i++){
		hipSetDevice(i);
		parTreeConstruction<<<1, 128>>>(list[i], world[i], worldSize, d_objs[i], numOBJs, d_nodes[i]);
		// worldGenerator<<<1,1>>>(list[i], world[i], worldSize, d_objs[i], numOBJs, 1);
		hipMalloc((void**)&d_img[i], sizeof(vec3)*x*y);
	}
	hipDeviceSynchronize();

	for(int i = 0; i < count; i++){
		hipSetDevice(i);
		for(int j = 0; j < numOBJs; j++){
			hipFree(h_d_nodes[i][j]);
		}
		delete[] h_d_nodes[i];
		hipFree(d_nodes[i]);
	}
	delete[] d_nodes;
	delete[] h_d_nodes;

	printf("Beginning Render\n");
	for(int i = 0; i < count; i++){
		hipSetDevice(i);
		imageGenerator<<<numBlocks, numThreads>>>(x, y, 1, cam, aaSamples/count, world[i], d_img[i], state[i]);//, d_hits[i], d_recs[i], d_dmax[i]);
		// getColor<<<numBlocks, numThreads>>>(x, y, aaSamples/count, cam, d_img[i], d_ray[i], world[i], state[i], color[i], hitRec[i], hits[i], cuRet[i]);//, d_hits[index], d_recs[index], d_dmax[index]);
		imgBuf[i] = new vec3[x*y];
	}

	hipDeviceSynchronize();
	printf("Done With Rendering, Copying to Disk/Cleaning\n");
	for(int i = 0; i < count; i++){
		hipSetDevice(i);
		clearWorld<<<1, 256>>>(world[i], 1);		
		hipMemcpy(imgBuf[i], d_img[i], sizeof(vec3)*x*y, hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		hipFree(state[i]);
		hipFree(world[i]);
		hipFree(list[i]);
		hipFree(d_img[i]);
	}
	printf("Done Cleaning, Merging from devices\n");
	delete[] state;
	delete[] world;
	delete[] list;
	delete[] d_img;

	hipSetDevice(count-1);
	hipDeviceSynchronize();
	
	vec3** d_imgs, **imgs;
	vec3* finImg, *img;
	// hipMalloc((void**)&d_imgs, sizeof(vec3)*x*y);
	// hipMemcpy(d_imgs, imgBuf, sizeof(vec3)*x*y, hipMemcpyHostToDevice);
	
	imgs = new vec3*[count];
	hipMalloc((void**)&d_imgs, count*sizeof(vec3*));
	hipMalloc((void**)&finImg, sizeof(vec3)*x*y);
	img = new vec3[x*y];

	for(int i = 0; i < count; i++){
		hipMalloc((void**)&imgs[i], x*y*sizeof(vec3));
	}
	hipDeviceSynchronize();
	for(int i = 0; i < count; i++){
		hipMemcpy(imgs[i], imgBuf[i], sizeof(vec3)*x*y, hipMemcpyHostToDevice);
	}
	hipMemcpy(d_imgs, imgs, count*sizeof(vec3*), hipMemcpyHostToDevice);
	
	float *d_r, *d_g, *d_b, *d_a;
	float *r, *g, *b, *a;
	hipMalloc((void**)&d_r, sizeof(float)*x*y);
	hipMalloc((void**)&d_g, sizeof(float)*x*y);
	hipMalloc((void**)&d_b, sizeof(float)*x*y);
	hipMalloc((void**)&d_a, sizeof(float)*x*y);
	hipDeviceSynchronize();

	// averageImgs<<<4, 512>>>(d_imgs, x, y, d_r, d_g, d_b, d_a);
	// __global__ void averageImgs(vec3* fin, vec3** img1, int count, int x, int y, float* r, float* g, float* b, float* a){
	averageImgs<<<1,1024>>>(finImg, d_imgs, count, x, y, d_r, d_g, d_b, d_a);
	r = new float[x*y];
	g = new float[x*y];
	b = new float[x*y];
	a = new float[x*y];
	hipDeviceSynchronize();

	// hipMemcpy(imgBuf, imgBuf, sizeof(vec3)*x*y, hipMemcpyDeviceToHost);
	hipMemcpy(r, d_r, sizeof(float)*x*y, hipMemcpyDeviceToHost);
	hipMemcpy(g, d_g, sizeof(float)*x*y, hipMemcpyDeviceToHost);
	hipMemcpy(b, d_b, sizeof(float)*x*y, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	hipFree(d_r);
	hipFree(d_g);
	hipFree(d_b);
	hipFree(d_a);
	// hipFree(d_imgs);
	// hipFree(finImg);
	// delete[] imgs;

	Header header(x, y);
	header.channels().insert("R", Channel(FLOAT));
	header.channels().insert("G", Channel(FLOAT));
	header.channels().insert("B", Channel(FLOAT));
	header.channels().insert("A", Channel(FLOAT));

	OutputFile file("out.exr", header);

	FrameBuffer frameBuffer;
	frameBuffer.insert("R", Slice(FLOAT, (char*)r, sizeof(*r)*1, sizeof(*r)*x));
	frameBuffer.insert("G", Slice(FLOAT, (char*)g, sizeof(*g)*1, sizeof(*g)*x));
	frameBuffer.insert("B", Slice(FLOAT, (char*)b, sizeof(*b)*1, sizeof(*b)*x));
	frameBuffer.insert("A", Slice(FLOAT, (char*)a, sizeof(*a)*1, sizeof(*a)*x));
	file.setFrameBuffer(frameBuffer);
	file.writePixels(y);

	delete[] r;
	delete[] g;
	delete[] b;
	delete[] a;
	// delete[] img;
	return 0;
}